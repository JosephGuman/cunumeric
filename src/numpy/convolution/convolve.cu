#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include <hip/hip_cooperative_groups.h>

#include "numpy/divmod.h"
#include "numpy/cuda_help.h"
#include "numpy/convolution/convolve.h"
#include "numpy/convolution/convolve_template.inl"

namespace legate {
namespace numpy {

using namespace Legion;

// For optimizing bandwidth utilization for convolution we load data
// from the input into shared memory and leave the filter in global memory
// with the expectation that it can be cached in the L2 and likely even
// the L1 cache of each SM across all threads and threadblocks. We stream
// reads from the inputs and stores from the outputs with the appropriate
// cache qualifiers in order to avoid polluting the filter data in the caches.

// We have several different variants of the convolution kernel to try to 
// minimize how often we load data. We do this by computing a "logical" tiling
// of the space. We want our logical tiling to be at least as wide as
// the filter in all dimensions so that we can minimize the amount of redundant
// data movement that needs to be done to perform the computation. We also require
// that the last dimension be loading at least contiguous bytes so we can get 
// coalesced loads. We begin by computing the logical tiling and seeing how
// much shared memory it requires:
// Case 1: The tiling requires less that SMEM_PER_CTA so we can fit the entire
//         computation in a threadblock. This is the nicest case and the one
//         that should result in the best performance since we'll be able to
//         load the data into shared memory and then have the threads loop
//         over all the points and compute their convolutions
// Case 2: We couldn't fit the whole tile in shared memory, so let's go for
//         the L2 cache. See if the tile fits in the L2 cache, if so grow the
//         tile up to 75% of the L2 cache size and launch a cooperative group
//         kernel to perform each tile across all the threads in the GPU, sync
//         and then move on to the next tile.
// Case 3: The whole tile couldn't fit in the L2, so pick a subset of the tile
//         that fits in the L2. See if the aggregate data for walking in the 
//         remaining dimensions can fit in the register files of all the SMs
//         in the GPU. If so we can grow the tile size until we hit either
//         75% of the L2 cache or we exhaust the register budget (depends on
//         the size of the untiled dimensions). Launch a cooperative group
//         kernel to iterate the tiles and sync between them to maintain
//         some degree of coherence in the L2 cache.
// Case 4: Either we don't support cooperative launches or this is truly
//         awful convolution and there is no hope for blocking it for 
//         on-chip memory in a reasonable way, so just give each thread
//         a point to compute and hope the cache gods are kind to you.

template<int DIM>
struct ConvolutionCase1Args {
  FastDivmodU64 grid_pitches[DIM];
  FastDivmodU64 block_pitches[DIM];
  FastDivmodU64 input_pitches[DIM];
  unsigned block_tiles[DIM];
  unsigned filter_centers[DIM];
  unsigned filter_extents[DIM];
  Point<DIM> delta_lo, delta_hi;
  size_t filter_volume;
  size_t tile_volume;
  size_t input_volume;
};

template<typename VAL, int DIM>
__global__ static void __launch_bounds__(512, 2)
convolution_case1a_kernel(const AccessorWO<VAL, DIM> out,
                          const AccessorRO<VAL, DIM> filter,
                          const AccessorRO<VAL, DIM> in,
                          const Rect<DIM> root_rect,
                          const Rect<DIM> subrect,
                          const Rect<DIM> filter_rect,
                          const ConvolutionCase1Args<DIM> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this illegal C++, but there's no other way to do it
  VAL *input = (VAL*)buffer;
  // Compute the origin point of the block
  size_t offset = blockIdx.x;
  Point<DIM> block_point = subrect.lo;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    block_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  // Load in the shared memory for this block
  Point<DIM> tile_point;
  const Rect<DIM> input_bounds(block_point - args.delta_lo, block_point + args.delta_hi);
  const bool input_contained = root_rect.contains(input_bounds);
  if (input_contained) {
    // All the points are contained, so no need for point-wise tests
    // Unroll this four times to try to pipeline loads
    #pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        tile_point[d] = args.input_pitches[d].divmod(offset,offset);
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  } else {
    // Need to do point-wise tests
    // Unroll this four times to try to pipeline loads
    #pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        tile_point[d] = args.input_pitches[d].divmod(offset,offset);
      if (!root_rect.contains(input_bounds.lo + tile_point))
        continue;
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  }
  // Wait for everything to be loaded into shared memory 
  __syncthreads();
  // Loop over points in the tile and compute the outputs
  coord_t f_coords[DIM];
  Point<DIM> out_point, in_point, filter_point;
  for (unsigned idx = threadIdx.x; idx < args.tile_volume; idx += blockDim.x) {
    // Compute the local coordinates
    offset = idx;
    #pragma unroll
    for (int d = 0; d < DIM; d++) {
      tile_point[d] = args.block_pitches[d].divmod(offset, offset); 
      out_point[d] = block_point[d] + tile_point[d];
    }
    if (!subrect.contains(out_point))
      continue;
    #pragma unroll
    for (int d = 0; d < DIM; d++)
      f_coords[d] = 0;
    VAL acc{0};
    for (unsigned idx = 0; idx < args.filter_volume; idx++) {
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
      if (input_contained || root_rect.contains(in_point))
      {
        offset = 0;
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          offset += (tile_point[d] + f_coords[d]) * args.input_pitches[d].divisor; 
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
        acc = acc + input[offset] * filter[filter_point];
      }
      // Step the filter coordinates
      #pragma unroll
      for (int d = DIM-1; d >= 0; d--) {
        f_coords[d]++;
        if (f_coords[d] == args.filter_extents[d])
          f_coords[d] = 0;
        else
          break;
      }
    }
    store_streaming(out.ptr(out_point), acc);
  }
}

// This version of the kernel is identical to the one above but with
// different launch bounds to handle a bigger CTA with more shared memory
template<typename VAL, int DIM>
__global__ static void __launch_bounds__(1024, 1)
convolution_case1b_kernel(const AccessorWO<VAL, DIM> out,
                          const AccessorRO<VAL, DIM> filter,
                          const AccessorRO<VAL, DIM> in,
                          const Rect<DIM> root_rect,
                          const Rect<DIM> subrect,
                          const Rect<DIM> filter_rect,
                          const ConvolutionCase1Args<DIM> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this illegal C++, but there's no other way to do it
  VAL *input = (VAL*)buffer;
  // Compute the origin point of the block
  size_t offset = blockIdx.x;
  Point<DIM> block_point = subrect.lo;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    block_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  // Load in the shared memory for this block
  Point<DIM> tile_point;
  const Rect<DIM> input_bounds(block_point - args.delta_lo, block_point + args.delta_hi);
  const bool input_contained = root_rect.contains(input_bounds);
  if (input_contained) {
    // All the points are contained, so no need for point-wise tests
    // Unroll this four times to try to pipeline loads
    #pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        tile_point[d] = args.input_pitches[d].divmod(offset,offset);
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  } else {
    // Need to do point-wise tests
    // Unroll this four times to try to pipeline loads
    #pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        tile_point[d] = args.input_pitches[d].divmod(offset,offset);
      if (!root_rect.contains(input_bounds.lo + tile_point))
        continue;
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  }
  // Wait for everything to be loaded into shared memory 
  __syncthreads();
  // Loop over points in the tile and compute the outputs
  coord_t f_coords[DIM];
  Point<DIM> out_point, in_point, filter_point;
  for (unsigned idx = threadIdx.x; idx < args.tile_volume; idx += blockDim.x) {
    // Compute the local coordinates
    offset = idx;
    #pragma unroll
    for (int d = 0; d < DIM; d++) {
      tile_point[d] = args.block_pitches[d].divmod(offset, offset); 
      out_point[d] = block_point[d] + tile_point[d];
    }
    if (!subrect.contains(out_point))
      continue;
    #pragma unroll
    for (int d = 0; d < DIM; d++)
      f_coords[d] = 0;
    VAL acc{0};
    for (unsigned idx = 0; idx < args.filter_volume; idx++) {
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
      if (input_contained || root_rect.contains(in_point))
      {
        offset = 0;
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          offset += (tile_point[d] + f_coords[d]) * args.input_pitches[d].divisor; 
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
        acc = acc + input[offset] * filter[filter_point];
      }
      // Step the filter coordinates
      #pragma unroll
      for (int d = DIM-1; d >= 0; d--) {
        f_coords[d]++;
        if (f_coords[d] == args.filter_extents[d])
          f_coords[d] = 0;
        else
          break;
      }
    }
    store_streaming(out.ptr(out_point), acc);
  }
}

template<int DIM>
struct ConvolutionCase2Args {
  FastDivmodU64 tile_pitches[DIM];
  size_t tile_strides[DIM];
  Point<DIM> delta_lo, delta_hi;
  unsigned filter_centers[DIM];
  unsigned filter_extents[DIM];
  unsigned filter_volume;
  unsigned thread_points;
  unsigned total_threads;
  unsigned tile_count;
};

template<typename VAL, int DIM>
__global__ static void __launch_bounds__(COOPERATIVE_THREADS,4)
convolution_case2_kernel(const AccessorWO<VAL, DIM> out,
                         const AccessorRO<VAL, DIM> filter,
                         const AccessorRO<VAL, DIM> in,
                         const Rect<DIM> root_rect,
                         const Rect<DIM> subrect,
                         const Rect<DIM> filter_rect,
                         const ConvolutionCase2Args<DIM> args)
{
  Point<DIM> tile_point = subrect.lo;
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  // Loop over the tiles
  for (unsigned tile = 0; tile < args.tile_count; tile++) {
    // Sync before each tile to make sure we aren't thrashing the L2
    if (tile > 0) 
      cooperative_groups::sync(cooperative_groups::this_grid());
    const Rect<DIM> input_bounds(tile_point - args.delta_lo, tile_point + args.delta_hi);
    const bool input_contained = root_rect.contains(input_bounds);
    // Loop over our output points and compute their convolutions
    for (unsigned point = 0; point < args.thread_points; point++) {
      // Compute our local point
      Point<DIM> out_point = tile_point;
      size_t offset = point * args.total_threads + tid;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        out_point[d] += args.tile_pitches[d].divmod(offset, offset);
      if (!subrect.contains(out_point))
        break;
      unsigned f_coords[DIM];
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        f_coords[d] = 0;
      VAL acc{0};
      Point<DIM> in_point, filter_point;
      for (unsigned idx = 0; idx < args.filter_volume; idx++) {
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
        if (input_contained || root_rect.contains(in_point))
        {
          #pragma unroll
          for (int d = 0; d < DIM; d++)
            filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
          // Only load inputs into the L2 cache with hope being that 
          // we'll be keeping the filter in the L1 cache or L2 cache
          acc = acc + load_l2(in.ptr(in_point)) * filter[filter_point];
        }
        // Step the filter coordinates
        #pragma unroll
        for (int d = DIM-1; d >= 0; d--) {
          f_coords[d]++;
          if (f_coords[d] == args.filter_extents[d])
            f_coords[d] = 0;
          else
            break;
        }
      }
      // Make sure the stores don't pollute the L2
      store_streaming(out.ptr(out_point), acc);
    }
    // Step to the next tile point
    for (int d = DIM-1; d >= 0; d--) {
      tile_point[d] += args.tile_strides[d];
      if (tile_point[d] > subrect.hi[d])
        tile_point[d] = subrect.lo[d];
      else
        break;
    }
  }
}

template<int DIM>
struct ConvolutionCase4Args {
  FastDivmodU64 grid_pitches[DIM];
  FastDivmodU64 block_pitches[DIM];
  unsigned block_tiles[DIM];
  unsigned filter_centers[DIM];
  unsigned filter_extents[DIM];
  size_t filter_volume;
};

template<typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 2)
convolution_case4_kernel(const AccessorWO<VAL, DIM> out,
                         const AccessorRO<VAL, DIM> filter,
                         const AccessorRO<VAL, DIM> in,
                         const Rect<DIM> root_rect,
                         const Rect<DIM> subrect,
                         const Rect<DIM> filter_rect,
                         const ConvolutionCase4Args<DIM> args)
{
  // Compute our local point from our block and thread IDs
  Point<DIM> out_point = subrect.lo;
  size_t offset = blockIdx.x;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    out_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  offset = threadIdx.x;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    out_point[d] += args.block_pitches[d].divmod(offset, offset);
  // If we're not computing an output there is nothing for us to do
  if (!subrect.contains(out_point))
    return;
  coord_t f_coords[DIM];
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    f_coords[d] = 0;
  VAL acc{0};
  Point<DIM> in_point, filter_point;
  for (unsigned idx = 0; idx < args.filter_volume; idx++) {
    #pragma unroll
    for (int d = 0; d < DIM; d++)
      in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
    if (root_rect.contains(in_point))
    {
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
      acc = acc + in[in_point] * filter[filter_point];
    }
    // Step the filter coordinates
    #pragma unroll
    for (int d = DIM-1; d >= 0; d--) {
      f_coords[d]++;
      if (f_coords[d] == args.filter_extents[d])
        f_coords[d] = 0;
      else
        break;
    }
  }
  store_streaming(out.ptr(out_point), acc);
}

template<typename VAL, int DIM>
__host__ static unsigned 
roundup_tile(unsigned tile[DIM],
             const unsigned centers[DIM],
             const unsigned max_size)
{
  if (DIM == 1) {
    // In this single case we can just solve for this directly
    unsigned elements = max_size / sizeof(VAL);
    assert(elements > 2*centers[0]);
    assert(tile[0] < (elements - 2*centers[0]));
    tile[0] = elements - 2*centers[0];
    return (tile[0] + 2*centers[0]) * sizeof(VAL);
  } else {
    // Find the two smallest dimensions and increase one of them
    // until we hit the second smallest one or exceed max_smem_size
    unsigned result = 0;
    bool all_same = true;
    while (true) {
      int d1 = DIM-1, d2 = -1;
      int t1 = tile[d1], t2 = 0;
      for (int d = DIM-2; d >= 0; d--) {
        if (tile[d] < t1) {
          d2 = d1;
          t2 = t1;
          d1 = d;
          t1 = tile[d];
        } else if ((d2 < 0) || (tile[d] < t2)) {
          d2 = d;
          t2 = tile[d];
        }
      }
      // If we ever get two dimensions of the same size then we know
      // that there is no smallest dimension so we can march all the
      // dimensions together at this point
      if (t1 == t2)
        break;
      // Solve for the max we can walk 
      unsigned pitch = sizeof(VAL);
      for (int d = 0; d < DIM; d++)
        if (d != d1)
          pitch *= (tile[d] + 2*centers[d]);
      unsigned elements = max_size / pitch;
      assert(elements > 2*centers[d1]);
      assert(t1 < (elements - 2*centers[d1]));
      unsigned bound = elements - 2*centers[d1];
      if (bound < t2) {
        tile[d1] = bound;
        result = pitch * (bound + 2*centers[d1]);
        all_same = false;
        break;
      } else {
        tile[d1] = t2;
        result = pitch * (t2 + 2*centers[d1]);
      }
    }
    if (all_same) {
      // Step all the dimensions together until we hit
      // the shared memory upper bound we're targetting
      // This algorithm is in theory slow, but the max
      // memory sizes of caches are "small" and the amount
      // of memory will grow polynomially in the number
      // of dimensions so it should converge quickly
      while (true) {
        unsigned next_size = sizeof(VAL);
        for (int d = 0; d < DIM; d++)
          next_size *= (tile[d] + 1 + 2*centers[d]);
        if (next_size > max_size) 
          break;
        result = next_size;
        for (int d = 0; d < DIM; d++)
          tile[d]++;
      }
    }
    return result;
  }
}

template <LegateTypeCode CODE, int DIM>
struct ConvolveImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  __host__
  void operator()(AccessorWO<VAL, DIM> out,
                  AccessorRO<VAL, DIM> filter,
                  AccessorRO<VAL, DIM> in,
                  const Rect<DIM>& root_rect,
                  const Rect<DIM>& subrect,
                  const Rect<DIM>& filter_rect) const
  {
    // Get the maximum amount of shared memory per threadblock
    int device;
    CHECK_CUDA( hipGetDevice(&device) );
    hipDeviceProp_t properties;
    CHECK_CUDA( hipGetDeviceProperties(&properties, device) );
    size_t max_smem_size = properties.sharedMemPerBlockOptin;

    unsigned extents[DIM];
    unsigned centers[DIM];
    for (int d = 0; d < DIM; d++) {
      assert(filter_rect.lo[d] == 0);
      extents[d] = filter_rect.hi[d] + 1;
      centers[d] = static_cast<coord_t>(extents[d] / 2);
    }
    unsigned tile[DIM];
    for (int d = DIM-1; d >= 0; d--) {
      // Make sure that each tile is at least double the size of the filter
      // so that we can get some savings in bandwidth needed 
      tile[d] = 2*centers[d];
      if (d == (DIM-1)) {
        // In order to maximize bandwidth, we want to make sure we're loading at
        // least 128B of contiguous memory along the last axis (row-major) of input
        const unsigned min_contig_elmts = 128 / sizeof(VAL);
        if ((tile[d] + 2*centers[d]) < min_contig_elmts)
          tile[d] = min_contig_elmts - 2*centers[d];
      } 
    }
    unsigned smem_size = sizeof(VAL);
    for (int d = 0; d < DIM; d++)
      smem_size *= (tile[d] + 2*centers[d]);
    if (smem_size <= max_smem_size) {
      // Case 1: Make the tile as big as possible so that it fits in shared memory
      // Try to keep it rectangular to minimize surface-to-volume ratio
      // and improve the reuse of data
      // If the current tile is less than half the shared memory in the SM then
      // decrease the upper bound so we can get 2 CTAs/SM
      bool halved = false;
      const unsigned half_smem = properties.sharedMemPerMultiprocessor / 2;
      if ((smem_size <= (half_smem)) && (half_smem < max_smem_size)) {
        max_smem_size = half_smem;
        halved = true;
      }
      smem_size = roundup_tile<VAL,DIM>(tile, centers, max_smem_size);
      // At this point we've got the tile size that we're going to compute
      // and the amount of dynamic shared memory that we need
      // Compute the arguments needed for the kernel launch
      ConvolutionCase1Args<DIM> args;
      size_t blocks = 1;
      size_t tile_pitch = 1;
      unsigned input_pitch = 1;
      args.filter_volume = 1;
      for (int d = DIM-1; d >= 0; d--) {
        size_t blocks_along_dim =
          ((subrect.hi[d] - subrect.lo[d]) + tile[d]) / tile[d];
        args.grid_pitches[d] = FastDivmodU64(blocks);
        blocks *= blocks_along_dim;
        args.block_tiles[d] = tile[d];
        args.block_pitches[d] = FastDivmodU64(tile_pitch);
        tile_pitch *= tile[d];
        args.delta_lo[d] = centers[d];
        args.delta_hi[d] = tile[d] + centers[d] - 1;
        args.input_pitches[d] = FastDivmodU64(input_pitch);
        input_pitch *= (args.delta_lo[d] + args.delta_hi[d] + 1);
        args.filter_centers[d] = centers[d]; 
        args.filter_extents[d] = extents[d];
        args.filter_volume *= extents[d];
      }
      args.tile_volume = tile_pitch;
      args.input_volume = input_pitch;
      assert((input_pitch * sizeof(VAL)) == smem_size);
      if (halved) {
        if (tile_pitch < 512)
          convolution_case1a_kernel<VAL,DIM><<<blocks,tile_pitch,smem_size>>>(
              out, filter, in, root_rect, subrect, filter_rect, args);
        else
          convolution_case1a_kernel<VAL,DIM><<<blocks,512,smem_size>>>(
              out, filter, in, root_rect, subrect, filter_rect, args);
      } else {
        if (tile_pitch < 1024)
          convolution_case1b_kernel<VAL,DIM><<<blocks,tile_pitch,smem_size>>>(
              out, filter, in, root_rect, subrect, filter_rect, args);
        else
          convolution_case1b_kernel<VAL,DIM><<<blocks,1024,smem_size>>>(
              out, filter, in, root_rect, subrect, filter_rect, args);
      }
      return;
    }
    // Check to see if we support cooperative launches
    if (properties.cooperativeLaunch) {
      // See if we fit in the L2 cache
      if (smem_size <= properties.l2CacheSize) {
        // Grow the tile to be at least 75% of L2 cache if it isn't already
        const unsigned threequartersl2 = 3 * properties.l2CacheSize / 4;
        if (smem_size < threequartersl2)
          roundup_tile<VAL,DIM>(tile, centers, threequartersl2);
        // Figure out how many blocks we can launch
        int blocksPerSM = 0;
        CHECK_CUDA( hipOccupancyMaxActiveBlocksPerMultiprocessor(&blocksPerSM,
              convolution_case2_kernel<VAL,DIM>, COOPERATIVE_THREADS, 0) );
        const size_t total_blocks = blocksPerSM * properties.multiProcessorCount;
        // Compute the arguments and launch the kernel
        ConvolutionCase2Args<DIM> args;
        size_t tile_pitch = 1;
        args.filter_volume = 1;
        args.tile_count = 1;
        for (int d = (DIM-1); d >= 0; d--) {
          args.tile_count *=
            (((subrect.hi[d] - subrect.lo[d]) + tile[d]) / tile[d]);
          args.tile_pitches[d] = FastDivmodU64(tile_pitch);
          tile_pitch *= tile[d];
          args.tile_strides[d] = tile[d];
          args.delta_lo[d] = centers[d];
          args.delta_hi[d] = tile[d] + centers[d] - 1;
          args.filter_centers[d] = centers[d]; 
          args.filter_extents[d] = extents[d];
          args.filter_volume *= extents[d];
        }
        size_t total_threads = total_blocks * COOPERATIVE_THREADS;
        args.thread_points = (tile_pitch + total_threads - 1) / total_threads;
        args.total_threads = total_threads;
        void *kernel_args[] =
          { (void*)&out, (void*)&filter, (void*)&in, (void*)&root_rect, 
            (void*)&subrect, (void*)&filter_rect, (void*)&args };
        CHECK_CUDA( hipLaunchCooperativeKernel((void*)convolution_case2_kernel<VAL,DIM>,
              total_blocks, COOPERATIVE_THREADS, kernel_args, 0/*null stream*/) );
        return;
      }
      // The whole tile doesn't fit in the L2 cache, see if we can 
      // find a subset that does while keeping all the partial 
      // convolution results for the remaining dimensions in the 
      // register files of all the SMs in the GPU
      

    }
    // Case 4: Either we don't support cooperative launches or this is just
    // a truly horrific convolution that it's just hopeless at trying to 
    // block for any of the on-chip memory so punt!
    // Figure out the tile size for the thread block. We want at
    // least 128B loads along the last dimension if possible. Then
    // round-robin powers of 2 onto the other dimensions until we 
    // get the tile to have as many threads as THREADS_PER_BLOCK.
    size_t limits[DIM];
    for (int d = 0; d < DIM; d++) {
      tile[d] = 1;
      limits[d] = subrect.hi[d] - subrect.lo[d] + 1;
    }
    // 2^5 == 32
    unsigned skip_dims = 0;
    for (int i = 0; i < 5; i++) {
      tile[DIM-1] *= 2;
      if (tile[DIM-1] >= limits[DIM-1]) {
        skip_dims |= (1 << (DIM-1));
        break;
      }
    }
    unsigned threads = tile[DIM-1];
    for (int i = 0; i < 5; i++) {
      for (int d = DIM-2; d >= 0; d--) {
        if (skip_dims & (1 << d))
          continue;
        tile[d] *= 2;
        threads *= 2;
        if (tile[d] >= limits[d]) {
          skip_dims |= (1 << d);
          continue;
        }
        if (threads == THREADS_PER_BLOCK)
          break;
      }
      if (threads == THREADS_PER_BLOCK)
        break;
    }
    while ((threads < THREADS_PER_BLOCK) &&
          (skip_dims != ((1 << (DIM+1)) - 1))) {
      for (int d = DIM-1; d >= 0; d--) {
        if (skip_dims & (1 << d))
          continue;
        tile[d] *= 2;
        threads *= 2;
        if (tile[d] >= limits[d]) {
          skip_dims |= (1 << d);
          continue;
        }
        if (threads == THREADS_PER_BLOCK)
          break;
      }
    }
    // should either not have enough points or
    // THREADS_PER_BLOCK should be a power of 2
    assert(threads <= THREADS_PER_BLOCK);
    // Compute the arguments needed to launch the kernel
    ConvolutionCase4Args<DIM> args;
    threads = 1;
    size_t blocks = 1;
    args.filter_volume = 1;
    for (int d = DIM-1; d >= 0; d--) {
      size_t blocks_along_dim =
        ((subrect.hi[d] - subrect.lo[d]) + tile[d]) / tile[d];
      args.grid_pitches[d] = FastDivmodU64(blocks);
      blocks *= blocks_along_dim;
      args.block_tiles[d] = tile[d];
      args.block_pitches[d] = FastDivmodU64(threads);
      threads *= tile[d];
      args.filter_centers[d] = centers[d]; 
      args.filter_extents[d] = extents[d];
      args.filter_volume *= extents[d];
    }
    convolution_case4_kernel<VAL,DIM><<<blocks,threads>>>(
        out, filter, in, root_rect, subrect, filter_rect, args);
  }
};

/*static*/ void ConvolveTask::gpu_variant(TaskContext& context)
{
  convolve_template<VariantKind::GPU>(context);
}

}  // namespace numpy
}  // namespace legate
