#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "numpy/divmod.h"
#include "numpy/cuda_help.h"
#include "numpy/convolution/convolve.h"
#include "numpy/convolution/convolve_template.inl"

namespace legate {
namespace numpy {

using namespace Legion;

// For optimizing bandwidth utilization for convolution we load data
// from the input into shared memory and leave the filter in global memory
// with the expectation that it can be cached in the L2 and likely even
// the L1 cache of each SM across all threads and threadblocks. We stream
// reads from the inputs and stores from the outputs with the appropriate
// cache qualifiers in order to avoid polluting the filter data in the caches.

// We have several different variants of the convolution kernel to try to 
// minimize how often we load data. We do this by computing a "logical" tiling
// of the space. We want our logical tiling to be at least as wide as
// the filter in all dimensions so that we can minimize the amount of redundant
// data movement that needs to be done to perform the computation. We also require
// that the last dimension be loading at least contiguous bytes so we can get 
// coalesced loads. We begin by computing the logical tiling and seeing how
// much shared memory it requires:
// Case 1: The tiling requires less that SMEM_PER_CTA so we can fit the entire
//         computation in a threadblock. This is the nicest case and the one
//         that should result in the best performance since we'll be able to
//         load the data into shared memory and then have the threads loop
//         over all the points and compute their convolutions
// Case 2: We couldn't fit the whole tile in shared memory, so let's go for
//         the L2 cache. See if the tile fits in the L2 cache, if so grow the
//         tile up to 75% of the L2 cache size and launch a cooperative group
//         kernel to perform each tile across all the threads in the GPU, sync
//         and then move on to the next tile.
// Case 3: The whole tile couldn't fit in the L2, so pick a subset of the tile
//         that fits in the L2. See if the aggregate data for walking in the 
//         remaining dimensions can fit in the register files of all the SMs
//         in the GPU. If so we can grow the tile size until we hit either
//         75% of the L2 cache or we exhaust the register budget (depends on
//         the size of the untiled dimensions). Launch a cooperative group
//         kernel to iterate the tiles and sync between them to maintain
//         some degree of coherence in the L2 cache.
// Case 4: Either we don't support cooperative launches or this is truly
//         awful convolution and there is no hope for blocking it for 
//         on-chip memory in a reasonable way, so just give each thread
//         a point to compute and hope the cache gods are kind to you.

template<int DIM>
struct ConvolutionInitArgs {
public:
  FastDivmodU64 pitches[DIM];
};

template<typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK,4)
convolution_init(const AccessorWO<VAL,DIM> out,
                 const Point<DIM> subrect_lo,
                 const ConvolutionInitArgs<DIM> args,
                 const size_t volume)
{
  size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= volume)
    return;
  Point<DIM> point = subrect_lo;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    point[d] += args.pitches[d].divmod(offset,offset);
  out[point] = VAL{0};
}

// We want to run the convolution kernel with as large a shared memory
// tile as possible to avoid duplicate loading of data and maximize 
// compute intensity. Therefore we're always going to run with 1 CTA
// per SM, but we still want enough thread-level parallelism, so we
// set this to the maximum number of warps in a threadblock
// Note that a lot of this code assumes this is a power of 2
#define CONVOLUTION_THREADS 1024
// The idea behind THREAD_OUTPUTS is to figure out how many registers
// we will be willing to assign to hold the partial output accumulations
// in each thread without using too many registers. Every GPU (with one
// exception) has 64K 32-bit registers per SM. We key off that and want
// to allocate a quarter of those registers for holding the partial accumulations
// We assume here that sizeof(VAL) is a power of 2
#define THREAD_OUTPUTS(TYPE) \
  ((4/*bytes/reg*/ * ((65536/4)/CONVOLUTION_THREADS)/*regs/thread*/) / sizeof(TYPE))

template<int DIM, int POINTS>
struct ConvolutionLargeTileArgs {
  FastDivmod l1_output_tile_pitches[DIM];
  FastDivmod l1_input_pitches[DIM];
  FastDivmod l1_filter_pitches[DIM];
  FastDivmod l1_output_pitches[DIM];
  Point<DIM,unsigned> point_offsets[POINTS];
  Point<DIM,unsigned> l2_output_tile;
  Point<DIM,unsigned> l2_filter_tile;
  Point<DIM,unsigned> l1_output_tile;
  Point<DIM,unsigned> l1_filter_tile;
  Point<DIM,unsigned> l2_output_limits;
  unsigned total_l2_outputs;
  unsigned total_l1_outputs;
  unsigned total_l1_filters;
  unsigned total_l1_points;
  unsigned l1_filter_points;
  unsigned l1_input_points;
  unsigned shared_input_offset;
  unsigned uniform_input_stride;
  unsigned shared_input_bound;
};

template<typename VAL, int DIM, int POINTS>
__global__ static void __launch_bounds__(CONVOLUTION_THREADS,1)
convolution_large_tile(const AccessorWO<VAL, DIM> out,
                       const AccessorRO<VAL, DIM> filter,
                       const AccessorRO<VAL, DIM> in,
                       const Rect<DIM> root_rect,
                       const Rect<DIM> subrect,
                       const Rect<DIM> l2_filter_rect,
                       const Point<DIM> l2_input_start,
                       const Point<DIM> l2_input_stop,
                       const Point<DIM> l1_input_start,
                       const Point<DIM,unsigned> zero,
                       const Point<DIM,unsigned> one,
                       const ConvolutionLargeTileArgs<DIM,POINTS> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this is illegal C++, but there's no other way to do it
  VAL *sharedmem = (VAL*)buffer;
  Point<DIM,unsigned> l2_output_offset = zero;
  for (unsigned l2_outidx = 0; l2_outidx < args.total_l2_outputs; l2_outidx++) {
    // Do a quick check here to see if all the inputs are contained for this tile
    // l2_input_start = subrect.lo + args.extents - l2_filter_rect.hi - one - l2_centers
    // l2_input_stop = subrect.lo + l2_output_tile - one + args.extents - l2_filter_rect.lo - one - l2_centers
    const Rect<DIM> l2_input_rect(l2_input_start + l2_output_offset,
                                  l2_input_stop + l2_output_offset);
    const bool input_contained = root_rect.contains(l2_input_rect);
    // Iterate the L1 output tiles that this threadblock should compute for the L2 output
    for (unsigned l1_outidx = blockIdx.x; 
          l1_outidx < args.total_l1_outputs; l1_outidx += gridDim.x) {
      Point<DIM,unsigned> l1_output_offset;  
      int offset = l1_outidx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        l1_output_offset[d] = 
          args.l1_output_tile_pitches[d].divmod(offset, offset) * args.l1_output_tile[d];
      // Handle the boundary case where an L1 tile is not contained in the L2 tile
      // becasue the L2 tile is overlapping a boundary. Note this decisions is the
      // same for all the threads in the threadblock so no bad divergence
      bool output_contained = true;
      #pragma unroll
      for (int d = 0; d < DIM; d++) {
        if ((subrect.lo[d] + (l2_output_offset[d] + l1_output_offset[d])) <= subrect.hi[d])
          continue;
        output_contained = false;
        break;
      }
      if (!output_contained)
        continue;
      // Initialize our point data
      VAL acc[POINTS];
      #pragma unroll
      for (int p = 0; p < POINTS; p++)
        acc[p] = VAL{0};
      // Iterate over the l1 filter tiles
      Point<DIM,unsigned> l1_filter_offset = zero;
      for (unsigned l1_fidx = 0; l1_fidx < args.total_l1_filters; l1_fidx++) {
        // Wait for any previous readers to be done
        __syncthreads();
        // Load the filter into shared memory
        // Unroll this a few times to get some memory level parallelims
        #pragma unroll 4
        for (unsigned fidx = threadIdx.x; fidx < args.l1_filter_points; fidx+=blockDim.x) {
          Point<DIM> filter_point = l2_filter_rect.lo + l1_filter_offset;
          offset = fidx;
          #pragma unroll
          for (int d = 0; d < DIM; d++)
            filter_point[d] += args.l1_filter_pitches[d].divmod(offset,offset);
          if (l2_filter_rect.contains(filter_point))
            sharedmem[fidx] = filter[filter_point];
          else
            sharedmem[fidx] = VAL{0};
        }
        // Load the input into shared memory
        // Compute the input start point
        // input_start = subrect.lo + extents - l2_filter_rect.lo - l1_filter_tile - l1_centers
        Point<DIM> input_start = l1_input_start + (l2_output_offset + l1_output_offset);
        input_start -= l1_filter_offset;
        // Unroll this a few times to get some memory level parallelism
        #pragma unroll 4
        for (unsigned idx = threadIdx.x; idx < args.l1_input_points; idx+=blockDim.x) {
          Point<DIM> input_point = input_start; 
          offset = idx;
          #pragma unroll
          for (int d = 0; d < DIM; d++)
            input_point[d] += args.l1_input_pitches[d].divmod(offset, offset);
          if (input_contained || root_rect.contains(input_point))
            sharedmem[args.shared_input_offset+idx] = in[input_point];
          else
            sharedmem[args.shared_input_offset+idx] = VAL{0};
        }
        // Wait for everything to be loaded into shared memory
        __syncthreads();
        // Iterate the points in the filter
        // We can safely iterate all the filter points and input points
        // because we wrote zeros into shared memory for everything that
        // was out of bounds
        Point<DIM,unsigned> filter_point = zero;
        for (unsigned fidx = 0; fidx < args.l1_filter_points; fidx++) {
          // Use shared memory broadcasting functionality to avoid bank conflicts
          const VAL filter_value = sharedmem[fidx]; 
          // Do the math for each of our points
          Point<DIM,unsigned> input_point = args.l1_filter_tile - filter_point - one;
          // There should never be any shared memory bank conflicts here since
          // the last dimension is indexed by a mod of threadIdx.x
          if (args.uniform_input_stride) {
            // Each point is a constant offset in shared from the others
            unsigned input_offset = args.shared_input_offset;
            offset = threadIdx.x;
            #pragma unroll
            for (int d = 0; d < DIM; d++)
              input_offset += args.l1_input_pitches[d].divisor *
                (input_point[d] + args.l1_output_pitches[d].divmod(offset, offset));
            if (args.shared_input_bound) {
              #pragma unroll
              for (int p = 0; p < POINTS; p++) {
                if (args.shared_input_bound <= input_offset)
                  break;
                acc[p] = acc[p] + filter_value * sharedmem[input_offset];
                input_offset += args.uniform_input_stride;
              }
            } else {
              #pragma unroll
              for (int p = 0; p < POINTS; p++) {
                acc[p] = acc[p] + filter_value * sharedmem[input_offset];
                input_offset += args.uniform_input_stride;
              }
            }
          } else {
            // Need to compute the input offset uniquely for each point
            Point<DIM,unsigned> offset_point;
            offset = threadIdx.x;
            #pragma unroll
            for (int d = 0; d < DIM; d++)
              offset_point[d] = args.l1_output_pitches[d].divmod(offset, offset);
            if (args.shared_input_bound) {
              #pragma unroll
              for (int p = 0; p < POINTS; p++) {
                unsigned input_offset = args.shared_input_offset;
                #pragma unroll
                for (int d = 0; d < DIM; d++)
                  input_offset += args.l1_input_pitches[d].divisor *
                      (input_point[d] + offset_point[d] + args.point_offsets[p][d]);
                if (args.shared_input_bound <= input_offset)
                  break;
                acc[p] = acc[p] + filter_value * sharedmem[input_offset];
              }
            } else {
              #pragma unroll
              for (int p = 0; p < POINTS; p++) {
                unsigned input_offset = args.shared_input_offset;
                #pragma unroll
                for (int d = 0; d < DIM; d++)
                  input_offset += args.l1_input_pitches[d].divisor *
                      (input_point[d] + offset_point[d] + args.point_offsets[p][d]);
                acc[p] = acc[p] + filter_value * sharedmem[input_offset];
              }
            }
          }
          // Step to the next filter point
          #pragma unroll
          for (int d = DIM-1; d >= 0; d--) {
            filter_point[d]++;
            if (filter_point[d] == args.l1_filter_tile[d])
              filter_point[d] = 0;
            else
              break;
          }
        }
        // Step to the next L1 filter tile
        #pragma unroll
        for (int d = DIM-1; d >= 0; d--) {
          l1_filter_offset[d] += args.l1_filter_tile[d];
          if (args.l2_filter_tile[d] <= l1_filter_offset[d])
            l1_filter_offset[d] = 0;
          else
            break;
        }
      }
      // Now we can stream our accumulators back to the output
      Point<DIM,unsigned> thread_offset;
      offset = threadIdx.x;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        thread_offset[d] = args.l1_output_pitches[d].divmod(offset, offset);
      Point<DIM> output = subrect.lo + (l2_output_offset + l1_output_offset + thread_offset);
      if (input_contained) {
        // If the input was contained, then so is the output
        if (args.total_l1_points) {
          unsigned index = threadIdx.x;
          #pragma unroll
          for (int p = 0; p < POINTS; p++) {
            if (args.total_l1_points <= index)
              break;
            VAL *ptr = out.ptr(output+args.point_offsets[p]);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
            index += blockDim.x;
          }
        } else {
          #pragma unroll
          for (int p = 0; p < POINTS; p++) {
            VAL *ptr = out.ptr(output+args.point_offsets[p]);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
          }
        }
      } else {
        // Input was not contained, so the output might not be either, do checks
        if (args.total_l1_points) {
          unsigned index = threadIdx.x;
          #pragma unroll
          for (int p = 0; p < POINTS; p++) {
            if (args.total_l1_points <= index)
              break;
            Point<DIM> point = output+args.point_offsets[p];
            if (!subrect.contains(point))
              break;
            VAL *ptr = out.ptr(point);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
            index += blockDim.x;
          }
        } else {
          #pragma unroll
          for (int p = 0; p < POINTS; p++) {
            Point<DIM> point = output+args.point_offsets[p];
            if (!subrect.contains(point))
              break;
            VAL *ptr = out.ptr(point);
            // Make sure we don't pollute the L2 cache
            VAL value = load_streaming<VAL>(ptr);
            store_streaming<VAL>(ptr, value + acc[p]);
          }
        }
      }
    }
    // Step to the next output tile
    #pragma unroll
    for (int d = DIM-1; d >= 0; d--) {
      l2_output_offset[d] += args.l2_output_tile[d];
      if (args.l2_output_limits[d] <= l2_output_offset[d])
        l2_output_offset[d] = 0;
      else
        break;
    }
  }
}

template<int DIM>
struct ConvolutionSmallTileArgs {
  FastDivmodU64 grid_pitches[DIM];
  FastDivmodU64 block_pitches[DIM];
  FastDivmodU64 input_pitches[DIM];
  unsigned block_tiles[DIM];
  unsigned filter_centers[DIM];
  unsigned filter_extents[DIM];
  Point<DIM> delta_lo, delta_hi;
  size_t filter_volume;
  size_t tile_volume;
  size_t input_volume;
};

template<typename VAL, int DIM>
__global__ static void __launch_bounds__(512, 2)
convolution_small_tile1(const AccessorWO<VAL, DIM> out,
                        const AccessorRO<VAL, DIM> filter,
                        const AccessorRO<VAL, DIM> in,
                        const Rect<DIM> root_rect,
                        const Rect<DIM> subrect,
                        const Rect<DIM> filter_rect,
                        const ConvolutionSmallTileArgs<DIM> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this illegal C++, but there's no other way to do it
  VAL *input = (VAL*)buffer;
  // Compute the origin point of the block
  size_t offset = blockIdx.x;
  Point<DIM> block_point = subrect.lo;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    block_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  // Load in the shared memory for this block
  Point<DIM> tile_point;
  const Rect<DIM> input_bounds(block_point - args.delta_lo, block_point + args.delta_hi);
  const bool input_contained = root_rect.contains(input_bounds);
  if (input_contained) {
    // All the points are contained, so no need for point-wise tests
    // Unroll this four times to try to pipeline loads
    #pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        tile_point[d] = args.input_pitches[d].divmod(offset,offset);
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  } else {
    // Need to do point-wise tests
    // Unroll this four times to try to pipeline loads
    #pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        tile_point[d] = args.input_pitches[d].divmod(offset,offset);
      if (!root_rect.contains(input_bounds.lo + tile_point))
        continue;
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  }
  // Wait for everything to be loaded into shared memory 
  __syncthreads();
  // Loop over points in the tile and compute the outputs
  coord_t f_coords[DIM];
  Point<DIM> out_point, in_point, filter_point;
  for (unsigned idx = threadIdx.x; idx < args.tile_volume; idx += blockDim.x) {
    // Compute the local coordinates
    offset = idx;
    #pragma unroll
    for (int d = 0; d < DIM; d++) {
      tile_point[d] = args.block_pitches[d].divmod(offset, offset); 
      out_point[d] = block_point[d] + tile_point[d];
    }
    if (!subrect.contains(out_point))
      continue;
    #pragma unroll
    for (int d = 0; d < DIM; d++)
      f_coords[d] = 0;
    VAL acc{0};
    for (unsigned idx = 0; idx < args.filter_volume; idx++) {
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
      if (input_contained || root_rect.contains(in_point))
      {
        offset = 0;
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          offset += (tile_point[d] + f_coords[d]) * args.input_pitches[d].divisor; 
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
        acc = acc + input[offset] * filter[filter_point];
      }
      // Step the filter coordinates
      #pragma unroll
      for (int d = DIM-1; d >= 0; d--) {
        f_coords[d]++;
        if (f_coords[d] == args.filter_extents[d])
          f_coords[d] = 0;
        else
          break;
      }
    }
    store_streaming(out.ptr(out_point), acc);
  }
}

// This version of the kernel is identical to the one above but with
// different launch bounds to handle a bigger CTA with more shared memory
template<typename VAL, int DIM>
__global__ static void __launch_bounds__(1024, 1)
convolution_small_tile2(const AccessorWO<VAL, DIM> out,
                        const AccessorRO<VAL, DIM> filter,
                        const AccessorRO<VAL, DIM> in,
                        const Rect<DIM> root_rect,
                        const Rect<DIM> subrect,
                        const Rect<DIM> filter_rect,
                        const ConvolutionSmallTileArgs<DIM> args)
{
  // Deal with compiler shared memory stupidity
  extern __shared__ uint8_t buffer[];
  // Technically this illegal C++, but there's no other way to do it
  VAL *input = (VAL*)buffer;
  // Compute the origin point of the block
  size_t offset = blockIdx.x;
  Point<DIM> block_point = subrect.lo;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    block_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  // Load in the shared memory for this block
  Point<DIM> tile_point;
  const Rect<DIM> input_bounds(block_point - args.delta_lo, block_point + args.delta_hi);
  const bool input_contained = root_rect.contains(input_bounds);
  if (input_contained) {
    // All the points are contained, so no need for point-wise tests
    // Unroll this four times to try to pipeline loads
    #pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        tile_point[d] = args.input_pitches[d].divmod(offset,offset);
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  } else {
    // Need to do point-wise tests
    // Unroll this four times to try to pipeline loads
    #pragma unroll 4
    for (unsigned idx = threadIdx.x; idx < args.input_volume; idx += blockDim.x) {
      offset = idx;
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        tile_point[d] = args.input_pitches[d].divmod(offset,offset);
      if (!root_rect.contains(input_bounds.lo + tile_point))
        continue;
      VAL value = in[input_bounds.lo + tile_point];
      // Write the value into shared memory
      input[idx] = value;
    }
  }
  // Wait for everything to be loaded into shared memory 
  __syncthreads();
  // Loop over points in the tile and compute the outputs
  coord_t f_coords[DIM];
  Point<DIM> out_point, in_point, filter_point;
  for (unsigned idx = threadIdx.x; idx < args.tile_volume; idx += blockDim.x) {
    // Compute the local coordinates
    offset = idx;
    #pragma unroll
    for (int d = 0; d < DIM; d++) {
      tile_point[d] = args.block_pitches[d].divmod(offset, offset); 
      out_point[d] = block_point[d] + tile_point[d];
    }
    if (!subrect.contains(out_point))
      continue;
    #pragma unroll
    for (int d = 0; d < DIM; d++)
      f_coords[d] = 0;
    VAL acc{0};
    for (unsigned idx = 0; idx < args.filter_volume; idx++) {
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
      if (input_contained || root_rect.contains(in_point))
      {
        offset = 0;
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          offset += (tile_point[d] + f_coords[d]) * args.input_pitches[d].divisor; 
        #pragma unroll
        for (int d = 0; d < DIM; d++)
          filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
        acc = acc + input[offset] * filter[filter_point];
      }
      // Step the filter coordinates
      #pragma unroll
      for (int d = DIM-1; d >= 0; d--) {
        f_coords[d]++;
        if (f_coords[d] == args.filter_extents[d])
          f_coords[d] = 0;
        else
          break;
      }
    }
    store_streaming(out.ptr(out_point), acc);
  }
}

#if 0
template<int DIM>
struct ConvolutionCase4Args {
  FastDivmodU64 grid_pitches[DIM];
  FastDivmodU64 block_pitches[DIM];
  unsigned block_tiles[DIM];
  unsigned filter_centers[DIM];
  unsigned filter_extents[DIM];
  size_t filter_volume;
};

template<typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, 2)
convolution_case4_kernel(const AccessorWO<VAL, DIM> out,
                         const AccessorRO<VAL, DIM> filter,
                         const AccessorRO<VAL, DIM> in,
                         const Rect<DIM> root_rect,
                         const Rect<DIM> subrect,
                         const Rect<DIM> filter_rect,
                         const ConvolutionCase4Args<DIM> args)
{
  // Compute our local point from our block and thread IDs
  Point<DIM> out_point = subrect.lo;
  size_t offset = blockIdx.x;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    out_point[d] += args.grid_pitches[d].divmod(offset, offset) * args.block_tiles[d];
  offset = threadIdx.x;
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    out_point[d] += args.block_pitches[d].divmod(offset, offset);
  // If we're not computing an output there is nothing for us to do
  if (!subrect.contains(out_point))
    return;
  coord_t f_coords[DIM];
  #pragma unroll
  for (int d = 0; d < DIM; d++)
    f_coords[d] = 0;
  VAL acc{0};
  Point<DIM> in_point, filter_point;
  for (unsigned idx = 0; idx < args.filter_volume; idx++) {
    #pragma unroll
    for (int d = 0; d < DIM; d++)
      in_point[d] = out_point[d] + f_coords[d] - args.filter_centers[d];
    if (root_rect.contains(in_point))
    {
      #pragma unroll
      for (int d = 0; d < DIM; d++)
        filter_point[d] = args.filter_extents[d] - f_coords[d] - 1;
      acc = acc + in[in_point] * filter[filter_point];
    }
    // Step the filter coordinates
    #pragma unroll
    for (int d = DIM-1; d >= 0; d--) {
      f_coords[d]++;
      if (f_coords[d] == args.filter_extents[d])
        f_coords[d] = 0;
      else
        break;
    }
  }
  store_streaming(out.ptr(out_point), acc);
}
#endif

template<typename VAL, int DIM>
__host__ static unsigned 
roundup_tile(unsigned tile[DIM],
             const unsigned centers[DIM],
             const unsigned max_size)
{
  if (DIM == 1) {
    // In this single case we can just solve for this directly
    unsigned elements = max_size / sizeof(VAL);
    assert(elements > 2*centers[0]);
    assert(tile[0] < (elements - 2*centers[0]));
    tile[0] = elements - 2*centers[0];
    return (tile[0] + 2*centers[0]) * sizeof(VAL);
  } else {
    // Find the two smallest dimensions and increase one of them
    // until we hit the second smallest one or exceed max_smem_size
    unsigned result = 0;
    bool all_same = true;
    while (true) {
      int d1 = DIM-1, d2 = -1;
      int t1 = tile[d1], t2 = 0;
      for (int d = DIM-2; d >= 0; d--) {
        if (tile[d] < t1) {
          d2 = d1;
          t2 = t1;
          d1 = d;
          t1 = tile[d];
        } else if ((d2 < 0) || (tile[d] < t2)) {
          d2 = d;
          t2 = tile[d];
        }
      }
      // If we ever get two dimensions of the same size then we know
      // that there is no smallest dimension so we can march all the
      // dimensions together at this point
      if (t1 == t2)
        break;
      // Solve for the max we can walk 
      unsigned pitch = sizeof(VAL);
      for (int d = 0; d < DIM; d++)
        if (d != d1)
          pitch *= (tile[d] + 2*centers[d]);
      unsigned elements = max_size / pitch;
      assert(elements > 2*centers[d1]);
      assert(t1 < (elements - 2*centers[d1]));
      unsigned bound = elements - 2*centers[d1];
      if (bound < t2) {
        tile[d1] = bound;
        result = pitch * (bound + 2*centers[d1]);
        all_same = false;
        break;
      } else {
        tile[d1] = t2;
        result = pitch * (t2 + 2*centers[d1]);
      }
    }
    if (all_same) {
      // Step all the dimensions together until we hit
      // the shared memory upper bound we're targetting
      // This algorithm is in theory slow, but the max
      // memory sizes of caches are "small" and the amount
      // of memory will grow polynomially in the number
      // of dimensions so it should converge quickly
      while (true) {
        unsigned next_size = sizeof(VAL);
        for (int d = 0; d < DIM; d++)
          next_size *= (tile[d] + 1 + 2*centers[d]);
        if (next_size > max_size) 
          break;
        result = next_size;
        for (int d = 0; d < DIM; d++)
          tile[d]++;
      }
    }
    return result;
  }
}


template <LegateTypeCode CODE, int DIM>
struct ConvolveImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  __host__
  void operator()(AccessorWO<VAL, DIM> out,
                  AccessorRO<VAL, DIM> filter,
                  AccessorRO<VAL, DIM> in,
                  const Rect<DIM>& root_rect,
                  const Rect<DIM>& subrect,
                  const Rect<DIM>& filter_rect) const
  {
    constexpr int THREADVALS = THREAD_OUTPUTS(VAL); 
    // Get the maximum amount of shared memory per threadblock
    int device;
    CHECK_CUDA( hipGetDevice(&device) );
    hipDeviceProp_t properties;
    CHECK_CUDA( hipGetDeviceProperties(&properties, device) );
    size_t max_smem_size = properties.sharedMemPerBlockOptin;

    // Only need to do these calls the first time on each device so 
    // we use a bit mask to track which devices we've done it for
    static unsigned long long mask = 0;
    if (!(mask & (1 << device))) {
      if (properties.sharedMemPerBlock < max_smem_size) {
        CHECK_CUDA( hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile1<VAL),DIM>,
              hipFuncAttributeMaxDynamicSharedMemorySize, max_smem_size) );
        CHECK_CUDA( hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_small_tile2<VAL),DIM>,
              hipFuncAttributeMaxDynamicSharedMemorySize, max_smem_size) );
        CHECK_CUDA( hipFuncSetAttribute(reinterpret_cast<const void*>(convolution_large_tile<VAL),DIM,THREADVALS>,
              hipFuncAttributeMaxDynamicSharedMemorySize, max_smem_size) );
      }
      if (sizeof(VAL) >= 8) {
      // Only need to set this on the first invocation
        CHECK_CUDA( hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(
              convolution_small_tile1<VAL),DIM>, hipSharedMemBankSizeEightByte) );
        CHECK_CUDA( hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(
              convolution_small_tile2<VAL),DIM>, hipSharedMemBankSizeEightByte) );
        CHECK_CUDA( hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(
              convolution_large_tile<VAL),DIM,THREADVALS>, hipSharedMemBankSizeEightByte) );
        
      }
      assert(device < 8*sizeof(mask));
      // Make sure not to race with updates from other GPUs
      __sync_fetch_and_add(&mask, (1 << device));
    }
    unsigned extents[DIM];
    unsigned centers[DIM];
    for (int d = 0; d < DIM; d++) {
      assert(filter_rect.lo[d] == 0);
      extents[d] = filter_rect.hi[d] + 1;
      centers[d] = static_cast<coord_t>(extents[d] / 2);
    }
#if 0
    unsigned tile[DIM];
    for (int d = DIM-1; d >= 0; d--) {
      // Make sure that each tile is at least double the size of the filter
      // so that we can get some savings in bandwidth needed 
      tile[d] = 2*centers[d];
      if (d == (DIM-1)) {
        // In order to maximize bandwidth, we want to make sure we're loading at
        // least 128B of contiguous memory along the last axis (row-major) of input
        const unsigned min_contig_elmts = 128 / sizeof(VAL);
        if ((tile[d] + 2*centers[d]) < min_contig_elmts)
          tile[d] = min_contig_elmts - 2*centers[d];
      } 
    }
    unsigned smem_size = sizeof(VAL);
    for (int d = 0; d < DIM; d++)
      smem_size *= (tile[d] + 2*centers[d]);
    if (smem_size <= max_smem_size) {
      // Small tile case:
      // Make the tile as big as possible so that it fits in shared memory
      // Try to keep it rectangular to minimize surface-to-volume ratio
      // and improve the reuse of data
      // If the current tile is less than half the shared memory in the SM then
      // decrease the upper bound so we can get 2 CTAs/SM
      bool halved = false;
      const unsigned half_smem = properties.sharedMemPerMultiprocessor / 2;
      if ((smem_size <= (half_smem)) && (half_smem < max_smem_size)) {
        max_smem_size = half_smem;
        halved = true;
      }
      smem_size = roundup_tile<VAL,DIM>(tile, centers, max_smem_size);
      // At this point we've got the tile size that we're going to compute
      // and the amount of dynamic shared memory that we need
      // Compute the arguments needed for the kernel launch
      ConvolutionSmallTileArgs<DIM> args;
      size_t blocks = 1;
      size_t tile_pitch = 1;
      unsigned input_pitch = 1;
      args.filter_volume = 1;
      for (int d = DIM-1; d >= 0; d--) {
        size_t blocks_along_dim =
          ((subrect.hi[d] - subrect.lo[d]) + tile[d]) / tile[d];
        args.grid_pitches[d] = FastDivmodU64(blocks);
        blocks *= blocks_along_dim;
        args.block_tiles[d] = tile[d];
        args.block_pitches[d] = FastDivmodU64(tile_pitch);
        tile_pitch *= tile[d];
        args.delta_lo[d] = centers[d];
        args.delta_hi[d] = tile[d] + centers[d] - 1;
        args.input_pitches[d] = FastDivmodU64(input_pitch);
        input_pitch *= (args.delta_lo[d] + args.delta_hi[d] + 1);
        args.filter_centers[d] = centers[d]; 
        args.filter_extents[d] = extents[d];
        args.filter_volume *= extents[d];
      }
      args.tile_volume = tile_pitch;
      args.input_volume = input_pitch;
      assert((input_pitch * sizeof(VAL)) == smem_size);
      if (halved) {
        if (tile_pitch < 512)
          convolution_small_tile1<VAL,DIM><<<blocks,tile_pitch,smem_size>>>(
              out, filter, in, root_rect, subrect, filter_rect, args);
        else
          convolution_small_tile1<VAL,DIM><<<blocks,512,smem_size>>>(
              out, filter, in, root_rect, subrect, filter_rect, args);
      } else {
        if (tile_pitch < 1024)
          convolution_small_tile2<VAL,DIM><<<blocks,tile_pitch,smem_size>>>(
              out, filter, in, root_rect, subrect, filter_rect, args);
        else
          convolution_small_tile2<VAL,DIM><<<blocks,1024,smem_size>>>(
              out, filter, in, root_rect, subrect, filter_rect, args);
      }
    } else {
#endif
    {
      // Large tile case:
      // If we're going to do this, we need to initialize the output to zeros
      // so we can kick that off to the GPU while we figure out how to launch
      // the rest of the kernels to do the convolution
      size_t strides[DIM];
      VAL *out_ptr = out.ptr(subrect, strides);
      // Check to see if the output is dense
      bool out_dense = true;
      size_t out_pitch = 1;
      for (int d = DIM-1; d >= 0; d--) {
        if (strides[d] != out_pitch) {
          out_dense = false;
          break;
        }
        out_pitch *= strides[d];
      }
      if (out_dense) {
        size_t bytes = sizeof(VAL) * subrect.volume();
        CHECK_CUDA( hipMemsetAsync(out_ptr, 0, bytes) );
      } else {
        out_pitch = 1;
        ConvolutionInitArgs<DIM> args;
        for (int d = DIM-1; d >= 0; d--) {
          args.pitches[d] = FastDivmodU64(out_pitch);
          out_pitch *= (subrect.hi[d] - subrect.lo[d] + 1);
        }
        size_t blocks = (out_pitch + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        convolution_init<VAL,DIM><<<blocks,THREADS_PER_BLOCK>>>(out, subrect.lo, args, out_pitch);
      }
      // Figure out the shape of the L1 output tile based on the number of
      // points that we can fit into registers
      Point<DIM> l1_output_tile;
      for (int d = 0; d < DIM; d++)
        l1_output_tile[d] = 1;
      unsigned max_l1_output_volume = CONVOLUTION_THREADS * THREADVALS;
      // Make sure the max_l1_output_volume doesn't consume more than half of shared memory
      bool need_shared_bound = false;
      while ((max_smem_size / 2) < (max_l1_output_volume * sizeof(VAL))) {
        max_l1_output_volume /= 2;
        need_shared_bound = true;
      }
      // Better be a power of 2
      assert((max_l1_output_volume & (max_l1_output_volume-1)) == 0);
      unsigned l1_output_volume = 1;
      // Try to make the last dimension at least 32 for coalescing
      const Point<DIM> output_bounds = subrect.hi - subrect.lo + Point<DIM>::ONES();
      for (int idx = 0; idx < 5; idx++) {
        l1_output_tile[DIM-1] *= 2;
        if (output_bounds[DIM-1] < l1_output_tile[DIM-1]) {
          l1_output_tile[DIM-1] /= 2;
          break;
        } else {
          l1_output_volume *= 2;
        }
        if (l1_output_volume == max_l1_output_volume)
          break;
      }
      int last_dim = DIM-1;
      // Round-robin powers of 2 onto the other dimensions until 
      // we hit the max or get all the dimensions balanced
      if (DIM > 1) {
        for (int idx = 0; idx < 5; idx++) {
          for (int d = DIM-2; d >= 0; d--) {
            l1_output_tile[d] *= 2;
            if (output_bounds[d] < l1_output_tile[d])
              l1_output_tile[d] /= 2;
            else {
              l1_output_volume *= 2;
              last_dim = d;
              if (l1_output_volume == max_l1_output_volume)
                break;
            }
          }
          if (l1_output_volume == max_l1_output_volume)
            break;
        }
      }
      // If we still have more to go round-robin powers of 2 over
      // all the dimensions
      int unchanged = 0;
      while (l1_output_volume < max_l1_output_volume) {
        if (last_dim == 0)
          last_dim = DIM-1;
        else
          last_dim--;
        l1_output_tile[last_dim] *= 2;
        if (output_bounds[last_dim] < l1_output_tile[last_dim]) {
          l1_output_tile[last_dim] /= 2;
          unchanged++;
          if (unchanged == DIM)
            break;
        } else {
          l1_output_volume *= 2;
          unchanged = 0;
        }
      }
      // At this point we've got our output tile, compute how big a filter
      // tile we can make and still fit both the filter tile and the 
      // input tile into the maximum amount of shared memory for this GPU
      Point<DIM> l1_filter_tile;
      for (int d = 0; d < DIM; d++)
        l1_filter_tile[d] = 1;
      const Point<DIM> filter_bounds = filter_rect.hi - filter_rect.lo + Point<DIM>::ONES();
      unsigned dynamic_smem = compute_filter_tile<VAL,DIM>(l1_filter_tile, 
                              filter_bounds, l1_output_tile, max_smem_size); 
      unsigned input_smem_offset = 1;
      for (int d = 0; d < DIM; d++)
        input_smem_offset *= l1_filter_tile[d];
      // Tile the number of SMs on this GPU to compute the shape of the 
      // L2 output tile for this kernel
      // We assume here that the number of SMs is easily factorable
      // into primes of 2, 3, and 5. It would be strange if we have a 
      // GPU with a number of SMs these days that can't be factored
      // this way. If we do report a warning.
      unsigned l2_tiles[DIM];
      for (int d = 0; d < DIM; d++)
        l2_tiles[d] = 1;
      if (DIM > 1) {
        unsigned twos = 0, threes = 0, fives = 0;
        unsigned remainder = properties.multiProcessorCount;
        while ((remainder > 1) && ((remainder % 2) == 0)) {
          twos++;
          remainder /= 2;
        }
        while ((remainder > 1) && ((remainder % 3) == 0)) {
          threes++;
          remainder /= 3;
        }
        while ((remainder > 1) && ((remainder % 5) == 0)) {
          fives++;
          remainder /= 5;
        }
        if (remainder > 1) {
          fprintf(stdout,"WARNING: %d is an unusual number of SMs "
              "for GPU convolution. Please report your GPU kind and "
              "the number of SMs in a Legate NumPy issue.",
              properties.multiProcessorCount);
          l2_tiles[DIM-1] = remainder;
        }
        for (unsigned idx = 0; idx < fives; idx++) {
          int smallest = 0;
          for (int d = 1; d < DIM; d++) {
            if (l2_tiles[smallest] < l2_tiles[d])
              continue;
            smallest = d;
          }
          l2_tiles[smallest] *= 5;
        }
        for (unsigned idx = 0; idx < threes; idx++) {
          int smallest = 0;
          for (int d = 1; d < DIM; d++) {
            if (l2_tiles[smallest] < l2_tiles[d])
              continue;
            smallest = d;
          }
          l2_tiles[smallest] *= 3;
        }
        for (unsigned idx = 0; idx < twos; idx++) {
          int smallest = 0;
          for (int d = 1; d < DIM; d++) {
            if (l2_tiles[smallest] < l2_tiles[d])
              continue;
            smallest = d;
          }
          l2_tiles[smallest] *= 2;
        }
      } else {
        l2_tiles[0] = properties.multiProcessorCount;
      }
      // Now that we've got a tiling of the l1 output blocks across
      // the SMs compute how big it is in memory and see if it is less
      // than a quarter of the L2 cache so we can block for the L2
      Point<DIM> l2_output_tile;
      size_t l2_output_tile_size = sizeof(VAL);
      for (int d = 0; d < DIM; d++) {
        l2_output_tile[d] = l2_tiles[d] * l1_output_tile[d];
        l2_output_tile_size *= l2_output_tile[d];
      }
      Point<DIM> l2_filter_tile; 
      size_t total_l2_filters = 1;
      if (l2_output_tile_size <= (properties.l2CacheSize/2)) {
        for (int d = 0; d < DIM; d++)
          l2_filter_tile[d] = 1;
        // Compute the L2 filter tile size so that the L2 filter and the 
        // corresponding L2 input tile will fit in the L2 cache
        compute_filter_tile<VAL,DIM>(l2_filter_tile, filter_bounds,
            l2_output_tile, properties.l2CacheSize);
        for (int d = 0; d < DIM; d++)
          total_l2_filters *= (filter_bounds[d] + l2_filter_tile[d] - 1) / l2_filter_tile[d];
      } else {
        // It's likely this tile is too big to block for the L2 cache
        // so we're not going to bother blocking for the L2 and just
        // run everything out of the framebuffer memory. The upside is
        // that we'll only need to make a single pass over the input
        for (int d = 0; d < DIM; d++)
          l2_filter_tile[d] = filter_rect.hi[d] - filter_rect.lo[d] + 1;
      }
      // Construct the arguments for the kernel launches
      ConvolutionLargeTileArgs<DIM,THREADVALS> args;
      int pitch = 1;
      for (int d = DIM-1; d >= 0; d--) {
        args.l1_input_pitches[d] = FastDivmod(pitch);
        pitch *= (l1_output_tile[d] + 2 * (l1_filter_tile[d]/2));
      }
      pitch = 1;
      for (int d = DIM-1; d >= 0; d--) {
        args.l1_filter_pitches[d] = FastDivmod(pitch);
        pitch *= l1_filter_tile[d];
      }
      pitch = 1;
      for (int d = DIM-1; d >= 0; d--) {
        args.l1_output_pitches[d] = FastDivmod(pitch);
        pitch *= l1_output_tile[d];
      }
      args.l2_output_tile = l2_output_tile;
      args.l2_filter_tile = l2_filter_tile;
      args.l1_output_tile = l1_output_tile;
      args.l1_filter_tile = l1_filter_tile;
      args.l2_output_limits = output_bounds;
      args.shared_input_offset = input_smem_offset; 
      args.total_l2_outputs = 1;
      args.total_l1_outputs = 1;
      args.total_l1_filters = 1;
      args.l1_filter_points = 1;
      args.l1_input_points = 1;
      pitch = 1;
      for (int d = DIM-1; d >= 0; d--) {
        args.total_l2_outputs *= (output_bounds[d] + l2_output_tile[d] - 1) / l2_output_tile[d];
        args.l1_output_tile_pitches[d] = FastDivmod(pitch);
        pitch *= (l2_output_tile[d] + l1_output_tile[d] - 1) / l1_output_tile[d];
        args.total_l1_filters *= (l2_filter_tile[d] + l1_filter_tile[d] - 1) / l1_filter_tile[d];
        args.l1_filter_points *= l1_filter_tile[d];
        args.l1_input_points *= (l1_output_tile[d] + 2 * (l1_filter_tile[d] / 2));
      }
      args.total_l1_outputs = pitch;
      // Figure out how to tile the points across the l1_output_tile
      if (DIM > 1) {
        unsigned regsteps[DIM];
        for (int d = 0; d < DIM; d++)
          regsteps[d] = 0;
        unsigned remainder = THREADVALS;
        for (int d = 0; d < DIM; d++) {
          if (remainder <= l1_output_tile[d]) {
            // All powers of two so should always divide
            assert((l1_output_tile[d] % remainder) == 0);
            regsteps[d] = l1_output_tile[d] / remainder;
            remainder = 1;
            break;
          } else {
            // All powers of two so should always divide
            assert((remainder % l1_output_tile[d]) == 0);
            regsteps[d] = 1;
            remainder /= l1_output_tile[d];
          }
        }
        assert(remainder == 1);
        Point<DIM,unsigned> offset = Point<DIM,unsigned>::ZEROES();
        for (int p = 0; p < THREADVALS; p++) {
          args.point_offsets[p] = offset;
          // Step to the next offset
          for (int d = DIM-1; d >= 0; d--) {
            offset[d] += regsteps[d];
            if (offset[d] == l1_output_tile[d])
              offset[d] = 0;
            else
              break;
          }
        }
        if ((regsteps[0] * THREADVALS) == l1_output_tile[0]) {
          args.uniform_input_stride = regsteps[0] * args.l1_input_pitches[0].divisor;
        } else {
          args.uniform_input_stride = 0;
        }
      } else {
        assert(THREADVALS <= l1_output_tile[0]);
        assert((l1_output_tile[0] % THREADVALS) == 0);
        unsigned regstep = l1_output_tile[0] / THREADVALS;
        for (int p = 0; p < THREADVALS; p++)
          args.point_offsets[p][0] = p * regstep;
        args.uniform_input_stride = regstep * args.l1_input_pitches[0].divisor;
      }
      if (need_shared_bound || (l1_output_volume < max_l1_output_volume)) {
        args.shared_input_bound = dynamic_smem / sizeof(VAL);
        args.total_l1_points = l1_output_volume;
      } else {
        args.shared_input_bound = 0;
        args.total_l1_points = 0;
      }
      // Launch as many kernels as we need to walk over the entire filter
      // Given the L2 filter tile that we came up with
      const Point<DIM,unsigned> zero = Point<DIM,unsigned>::ZEROES();
      const Point<DIM,unsigned> one = Point<DIM,unsigned>::ONES();
      Point<DIM> l1_centers;
      for (int d = 0; d < DIM; d++)
        l1_centers[d] = l1_filter_tile[d] / 2;
      if (total_l2_filters > 1) {
        Point<DIM> l2_filter_lo = filter_rect.lo;
        for (unsigned idx = 0; idx < total_l2_filters; idx++) {
          Rect<DIM> l2_filter_rect(l2_filter_lo, l2_filter_lo + l2_filter_tile - one);
          l2_filter_rect = l2_filter_rect.intersection(filter_rect);
          Point<DIM> l2_centers;
          for (int d = 0; d < DIM; d++)
            l2_centers[d] = (l2_filter_rect.hi[d] - l2_filter_rect.lo[d] + 1) / 2;
          const Point<DIM> l1_input_start = subrect.lo + Point<DIM>(extents) - l2_filter_lo - l1_filter_tile - l1_centers;
          const Point<DIM> l2_input_start = subrect.lo + Point<DIM>(extents) - l2_filter_rect.hi - one - l2_centers;
          const Point<DIM> l2_input_stop = subrect.lo + l2_output_tile - one + Point<DIM>(extents) - l2_filter_rect.lo - one - l2_centers;
          convolution_large_tile<VAL,DIM,THREADVALS><<<
            properties.multiProcessorCount,CONVOLUTION_THREADS,dynamic_smem>>>(
                out, filter, in, root_rect, subrect, l2_filter_rect, l2_input_start,
                l2_input_stop, l1_input_start, zero, one, args);
          // Step to the next filter
          for (int d = DIM-1; d >= 0; d--) {
            l2_filter_lo[d] += l2_filter_tile[d];
            if (filter_rect.hi[d] < l2_filter_lo[d])
              l2_filter_lo[d] = filter_rect.lo[d];
            else
              break;
          }
        }
      } else {
        assert(total_l2_filters == 1);
        const Point<DIM> l1_input_start = subrect.lo + Point<DIM>(extents) - filter_rect.lo - l1_filter_tile - l1_centers;
        const Point<DIM> l2_input_start = subrect.lo + Point<DIM>(extents) - filter_rect.lo - one - Point<DIM>(centers);
        const Point<DIM> l2_input_stop = subrect.lo + l2_output_tile - one + Point<DIM>(extents) - filter_rect.lo - one - Point<DIM>(centers);
        convolution_large_tile<VAL,DIM,THREADVALS><<<
          properties.multiProcessorCount,CONVOLUTION_THREADS,dynamic_smem>>>(
              out, filter, in, root_rect, subrect, filter_rect, l2_input_start,
              l2_input_stop, l1_input_start, zero, one, args);
      }
    }
#if 0
    // Case 4: Either we don't support cooperative launches or this is just
    // a truly horrific convolution that it's just hopeless at trying to 
    // block for any of the on-chip memory so punt!
    // Figure out the tile size for the thread block. We want at
    // least 128B loads along the last dimension if possible. Then
    // round-robin powers of 2 onto the other dimensions until we 
    // get the tile to have as many threads as THREADS_PER_BLOCK.
    size_t limits[DIM];
    for (int d = 0; d < DIM; d++) {
      tile[d] = 1;
      limits[d] = subrect.hi[d] - subrect.lo[d] + 1;
    }
    // 2^5 == 32
    unsigned skip_dims = 0;
    for (int i = 0; i < 5; i++) {
      tile[DIM-1] *= 2;
      if (tile[DIM-1] >= limits[DIM-1]) {
        skip_dims |= (1 << (DIM-1));
        break;
      }
    }
    unsigned threads = tile[DIM-1];
    for (int i = 0; i < 5; i++) {
      for (int d = DIM-2; d >= 0; d--) {
        if (skip_dims & (1 << d))
          continue;
        tile[d] *= 2;
        threads *= 2;
        if (tile[d] >= limits[d]) {
          skip_dims |= (1 << d);
          continue;
        }
        if (threads == THREADS_PER_BLOCK)
          break;
      }
      if (threads == THREADS_PER_BLOCK)
        break;
    }
    while ((threads < THREADS_PER_BLOCK) &&
          (skip_dims != ((1 << (DIM+1)) - 1))) {
      for (int d = DIM-1; d >= 0; d--) {
        if (skip_dims & (1 << d))
          continue;
        tile[d] *= 2;
        threads *= 2;
        if (tile[d] >= limits[d]) {
          skip_dims |= (1 << d);
          continue;
        }
        if (threads == THREADS_PER_BLOCK)
          break;
      }
    }
    // should either not have enough points or
    // THREADS_PER_BLOCK should be a power of 2
    assert(threads <= THREADS_PER_BLOCK);
    // Compute the arguments needed to launch the kernel
    ConvolutionCase4Args<DIM> args;
    threads = 1;
    size_t blocks = 1;
    args.filter_volume = 1;
    for (int d = DIM-1; d >= 0; d--) {
      size_t blocks_along_dim =
        ((subrect.hi[d] - subrect.lo[d]) + tile[d]) / tile[d];
      args.grid_pitches[d] = FastDivmodU64(blocks);
      blocks *= blocks_along_dim;
      args.block_tiles[d] = tile[d];
      args.block_pitches[d] = FastDivmodU64(threads);
      threads *= tile[d];
      args.filter_centers[d] = centers[d]; 
      args.filter_extents[d] = extents[d];
      args.filter_volume *= extents[d];
    }
    convolution_case4_kernel<VAL,DIM><<<blocks,threads>>>(
        out, filter, in, root_rect, subrect, filter_rect, args);
#endif
  }
};

/*static*/ void ConvolveTask::gpu_variant(TaskContext& context)
{
  convolve_template<VariantKind::GPU>(context);
}

}  // namespace numpy
}  // namespace legate
