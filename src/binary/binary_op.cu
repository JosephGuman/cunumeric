#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "binary/binary_op.h"
#include "core.h"
#include "dispatch.h"
#include "point_task.h"

#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

namespace legate {
namespace numpy {

using namespace Legion;

namespace gpu {

template <typename Function, typename ARG, typename RES>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  dense_kernel(size_t volume, Function func, RES *out, const ARG *in1, const ARG *in2)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  out[idx] = func(in1[idx], in2[idx]);
}

template <typename Function, typename ReadAcc, typename WriteAcc, typename Pitches, typename Rect>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM) generic_kernel(
  size_t volume, Function func, WriteAcc out, ReadAcc in1, ReadAcc in2, Pitches pitches, Rect rect)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, rect.lo);
  out[point] = func(in1[point], in2[point]);
}

template <BinaryOpCode OP_CODE>
struct BinaryOpImpl {
  template <LegateTypeCode CODE,
            int DIM,
            std::enable_if_t<BinaryOp<OP_CODE, CODE>::valid> * = nullptr>
  void operator()(Shape &shape, RegionField &out_rf, RegionField &in1_rf, RegionField &in2_rf)
  {
    using OP  = BinaryOp<OP_CODE, CODE>;
    using ARG = legate_type_of<CODE>;
    using RES = std::result_of_t<OP(ARG, ARG)>;

    auto rect = shape.to_rect<DIM>();

    Pitches<DIM - 1> pitches;
    size_t volume = pitches.flatten(rect);

    if (volume == 0) return;

    auto out = out_rf.write_accessor<RES, DIM>();
    auto in1 = in1_rf.read_accessor<ARG, DIM>();
    auto in2 = in2_rf.read_accessor<ARG, DIM>();

#ifndef LEGION_BOUNDS_CHECKS
    // Check to see if this is dense or not
    bool dense = out.accessor.is_dense_row_major(rect) && in1.accessor.is_dense_row_major(rect) &&
                 in2.accessor.is_dense_row_major(rect);
#else
    // No dense execution if we're doing bounds checks
    bool dense = false;
#endif

    OP func{};
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    if (dense) {
      auto outptr = out.ptr(rect);
      auto in1ptr = in1.ptr(rect);
      auto in2ptr = in2.ptr(rect);
      dense_kernel<<<blocks, THREADS_PER_BLOCK>>>(volume, func, outptr, in1ptr, in2ptr);
    } else {
      generic_kernel<<<blocks, THREADS_PER_BLOCK>>>(volume, func, out, in1, in2, pitches, rect);
    }
  }

  template <LegateTypeCode CODE,
            int DIM,
            std::enable_if_t<!BinaryOp<OP_CODE, CODE>::valid> * = nullptr>
  void operator()(Shape &shape, RegionField &out_rf, RegionField &in1_rf, RegionField &in2_rf)
  {
    assert(false);
  }
};

struct BinaryOpDispatch {
  template <BinaryOpCode OP_CODE>
  void operator()(Shape &shape, RegionField &out, RegionField &in1, RegionField &in2)
  {
    double_dispatch(in1.dim(), in1.code(), BinaryOpImpl<OP_CODE>{}, shape, out, in1, in2);
  }
};

}  // namespace gpu

/*static*/ void BinaryOpTask::gpu_variant(const Task *task,
                                          const std::vector<PhysicalRegion> &regions,
                                          Context context,
                                          Runtime *runtime)
{
  Deserializer ctx(task, regions);

  BinaryOpCode op_code;
  Shape shape;
  RegionField out;
  RegionField in1;
  RegionField in2;

  deserialize(ctx, op_code);
  deserialize(ctx, shape);
  deserialize(ctx, out);
  deserialize(ctx, in1);
  deserialize(ctx, in2);

  op_dispatch(op_code, gpu::BinaryOpDispatch{}, shape, out, in1, in2);
}

}  // namespace numpy
}  // namespace legate
