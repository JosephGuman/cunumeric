#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "arccos.h"

namespace legate {
namespace numpy {
// instantiate ArcCos' tasks' gpu variants
template void ArcCos<__half>::instantiate_task_gpu_variants();
template void ArcCos<float>::instantiate_task_gpu_variants();
template void ArcCos<double>::instantiate_task_gpu_variants();
template void ArcCos<int16_t>::instantiate_task_gpu_variants();
template void ArcCos<int32_t>::instantiate_task_gpu_variants();
template void ArcCos<int64_t>::instantiate_task_gpu_variants();
template void ArcCos<uint16_t>::instantiate_task_gpu_variants();
template void ArcCos<uint32_t>::instantiate_task_gpu_variants();
template void ArcCos<uint64_t>::instantiate_task_gpu_variants();
template void ArcCos<bool>::instantiate_task_gpu_variants();
template void ArcCos<complex<float>>::instantiate_task_gpu_variants();
template void ArcCos<complex<double>>::instantiate_task_gpu_variants();
}    // namespace numpy
}    // namespace legate
