#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "logical_not.h"

// instantiate LogicalNot's tasks' gpu variants
namespace legate {
namespace numpy {
template void LogicalNot<__half>::instantiate_task_gpu_variants();
template void LogicalNot<float>::instantiate_task_gpu_variants();
template void LogicalNot<double>::instantiate_task_gpu_variants();
template void LogicalNot<int16_t>::instantiate_task_gpu_variants();
template void LogicalNot<int32_t>::instantiate_task_gpu_variants();
template void LogicalNot<int64_t>::instantiate_task_gpu_variants();
template void LogicalNot<uint16_t>::instantiate_task_gpu_variants();
template void LogicalNot<uint32_t>::instantiate_task_gpu_variants();
template void LogicalNot<uint64_t>::instantiate_task_gpu_variants();
template void LogicalNot<bool>::instantiate_task_gpu_variants();
template void LogicalNot<complex<float>>::instantiate_task_gpu_variants();
template void LogicalNot<complex<double>>::instantiate_task_gpu_variants();
}  // namespace numpy
}  // namespace legate
