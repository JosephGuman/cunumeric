#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "sqrt.h"

// instantiate Sqrt's tasks' gpu variants
namespace legate {
namespace numpy {
template void Sqrt<__half>::instantiate_task_gpu_variants();
template void Sqrt<float>::instantiate_task_gpu_variants();
template void Sqrt<double>::instantiate_task_gpu_variants();
template void Sqrt<int16_t>::instantiate_task_gpu_variants();
template void Sqrt<int32_t>::instantiate_task_gpu_variants();
template void Sqrt<int64_t>::instantiate_task_gpu_variants();
template void Sqrt<uint16_t>::instantiate_task_gpu_variants();
template void Sqrt<uint32_t>::instantiate_task_gpu_variants();
template void Sqrt<uint64_t>::instantiate_task_gpu_variants();
template void Sqrt<bool>::instantiate_task_gpu_variants();
template void Sqrt<complex<float>>::instantiate_task_gpu_variants();
template void Sqrt<complex<double>>::instantiate_task_gpu_variants();
}    // namespace numpy
}    // namespace legate
