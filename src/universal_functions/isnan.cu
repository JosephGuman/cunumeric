#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "isnan.h"

// instantiate IsNaN's tasks' gpu variants
namespace legate {
namespace numpy {
template void IsNaN<__half>::instantiate_task_gpu_variants();
template void IsNaN<float>::instantiate_task_gpu_variants();
template void IsNaN<double>::instantiate_task_gpu_variants();
template void IsNaN<int16_t>::instantiate_task_gpu_variants();
template void IsNaN<int32_t>::instantiate_task_gpu_variants();
template void IsNaN<int64_t>::instantiate_task_gpu_variants();
template void IsNaN<uint16_t>::instantiate_task_gpu_variants();
template void IsNaN<uint32_t>::instantiate_task_gpu_variants();
template void IsNaN<uint64_t>::instantiate_task_gpu_variants();
template void IsNaN<bool>::instantiate_task_gpu_variants();
template void IsNaN<complex<float>>::instantiate_task_gpu_variants();
template void IsNaN<complex<double>>::instantiate_task_gpu_variants();
}    // namespace numpy
}    // namespace legate
