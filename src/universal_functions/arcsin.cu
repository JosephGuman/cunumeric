#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "arcsin.h"

namespace legate {
namespace numpy {
// instantiate ArcSin's tasks' gpu variants
template void ArcSin<__half>::instantiate_task_gpu_variants();
template void ArcSin<float>::instantiate_task_gpu_variants();
template void ArcSin<double>::instantiate_task_gpu_variants();
template void ArcSin<int16_t>::instantiate_task_gpu_variants();
template void ArcSin<int32_t>::instantiate_task_gpu_variants();
template void ArcSin<int64_t>::instantiate_task_gpu_variants();
template void ArcSin<uint16_t>::instantiate_task_gpu_variants();
template void ArcSin<uint32_t>::instantiate_task_gpu_variants();
template void ArcSin<uint64_t>::instantiate_task_gpu_variants();
template void ArcSin<bool>::instantiate_task_gpu_variants();
template void ArcSin<complex<float>>::instantiate_task_gpu_variants();
template void ArcSin<complex<double>>::instantiate_task_gpu_variants();
}  // namespace numpy
}  // namespace legate
