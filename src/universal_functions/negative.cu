#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "negative.h"

// instantiate Negative's tasks' gpu variants
namespace legate {
namespace numpy {
template void Negative<__half>::instantiate_task_gpu_variants();
template void Negative<float>::instantiate_task_gpu_variants();
template void Negative<double>::instantiate_task_gpu_variants();
template void Negative<int16_t>::instantiate_task_gpu_variants();
template void Negative<int32_t>::instantiate_task_gpu_variants();
template void Negative<int64_t>::instantiate_task_gpu_variants();
template void Negative<uint16_t>::instantiate_task_gpu_variants();
template void Negative<uint32_t>::instantiate_task_gpu_variants();
template void Negative<uint64_t>::instantiate_task_gpu_variants();
template void Negative<bool>::instantiate_task_gpu_variants();
template void Negative<complex<float>>::instantiate_task_gpu_variants();
template void Negative<complex<double>>::instantiate_task_gpu_variants();
}    // namespace numpy
}    // namespace legate
