#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "exp.h"

namespace legate {
namespace numpy {
// Instantiate Exp's tasks' gpu variants
template void Exp<__half>::instantiate_task_gpu_variants();
template void Exp<float>::instantiate_task_gpu_variants();
template void Exp<double>::instantiate_task_gpu_variants();
template void Exp<int16_t>::instantiate_task_gpu_variants();
template void Exp<int32_t>::instantiate_task_gpu_variants();
template void Exp<int64_t>::instantiate_task_gpu_variants();
template void Exp<uint16_t>::instantiate_task_gpu_variants();
template void Exp<uint32_t>::instantiate_task_gpu_variants();
template void Exp<uint64_t>::instantiate_task_gpu_variants();
template void Exp<bool>::instantiate_task_gpu_variants();
template void Exp<complex<float>>::instantiate_task_gpu_variants();
template void Exp<complex<double>>::instantiate_task_gpu_variants();
}  // namespace numpy
}  // namespace legate
