#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "tan.h"

// instantiate Tan's tasks' gpu variants
namespace legate {
namespace numpy {
template void Tan<__half>::instantiate_task_gpu_variants();
template void Tan<float>::instantiate_task_gpu_variants();
template void Tan<double>::instantiate_task_gpu_variants();
template void Tan<int16_t>::instantiate_task_gpu_variants();
template void Tan<int32_t>::instantiate_task_gpu_variants();
template void Tan<int64_t>::instantiate_task_gpu_variants();
template void Tan<uint16_t>::instantiate_task_gpu_variants();
template void Tan<uint32_t>::instantiate_task_gpu_variants();
template void Tan<uint64_t>::instantiate_task_gpu_variants();
template void Tan<bool>::instantiate_task_gpu_variants();
template void Tan<complex<float>>::instantiate_task_gpu_variants();
template void Tan<complex<double>>::instantiate_task_gpu_variants();
}    // namespace numpy
}    // namespace legate
