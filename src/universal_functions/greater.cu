#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "greater.h"

// Instantiate Greater's tasks' gpu variants
namespace legate {
namespace numpy {
template void Greater<__half>::instantiate_task_gpu_variants();
template void Greater<float>::instantiate_task_gpu_variants();
template void Greater<double>::instantiate_task_gpu_variants();
template void Greater<int16_t>::instantiate_task_gpu_variants();
template void Greater<int32_t>::instantiate_task_gpu_variants();
template void Greater<int64_t>::instantiate_task_gpu_variants();
template void Greater<uint16_t>::instantiate_task_gpu_variants();
template void Greater<uint32_t>::instantiate_task_gpu_variants();
template void Greater<uint64_t>::instantiate_task_gpu_variants();
template void Greater<bool>::instantiate_task_gpu_variants();
template void Greater<complex<float>>::instantiate_task_gpu_variants();
template void Greater<complex<double>>::instantiate_task_gpu_variants();
}    // namespace numpy
}    // namespace legate
