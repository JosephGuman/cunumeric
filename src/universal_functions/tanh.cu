#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "tanh.h"

// instantiate Tanh's tasks' gpu variants
namespace legate {
namespace numpy {
template void Tanh<__half>::instantiate_task_gpu_variants();
template void Tanh<float>::instantiate_task_gpu_variants();
template void Tanh<double>::instantiate_task_gpu_variants();
template void Tanh<int16_t>::instantiate_task_gpu_variants();
template void Tanh<int32_t>::instantiate_task_gpu_variants();
template void Tanh<int64_t>::instantiate_task_gpu_variants();
template void Tanh<uint16_t>::instantiate_task_gpu_variants();
template void Tanh<uint32_t>::instantiate_task_gpu_variants();
template void Tanh<uint64_t>::instantiate_task_gpu_variants();
template void Tanh<bool>::instantiate_task_gpu_variants();
template void Tanh<complex<float>>::instantiate_task_gpu_variants();
template void Tanh<complex<double>>::instantiate_task_gpu_variants();
}    // namespace numpy
}    // namespace legate
