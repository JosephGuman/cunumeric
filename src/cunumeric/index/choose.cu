#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/index/choose.h"
#include "cunumeric/index/choose_template.inl"
#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;

template <typename VAL, int DIM>
__global__ static void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  choose_from_tuple_kernel(const AccessorWO<VAL, DIM> out,
                           const AccessorRO<int, DIM> index_arr,
                           const AccessorRO<VAL, DIM>* choices,
                           const Rect<DIM> rect,
                           const Pitches<DIM - 1> pitches,
                           int volume)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto p = pitches.unflatten(idx, rect.lo);
  out[p] = choices[index_arr[p]][p];
}

template <LegateTypeCode CODE, int DIM>
struct ChooseImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  void operator()(const AccessorWO<VAL, DIM>& out,
                  const AccessorRO<int, DIM>& index_arr,
                  const std::vector<AccessorRO<VAL, DIM>>& choices,
                  const Rect<DIM>& rect,
                  const Pitches<DIM - 1>& pitches,
                  bool dense) const
  {
    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    AccessorRO<VAL, DIM>* ch_arr;
    hipMalloc((void**)&ch_arr, choices.size() * sizeof(AccessorRO<VAL, DIM>));
    hipMemcpy(ch_arr,
               choices.data(),
               choices.size() * sizeof(AccessorRO<VAL, DIM>),
               hipMemcpyHostToDevice);
    choose_from_tuple_kernel<VAL, DIM>
      <<<blocks, THREADS_PER_BLOCK>>>(out, index_arr, ch_arr, rect, pitches, volume);
    hipFree(ch_arr);
  }
};

/*static*/ void ChooseTask::gpu_variant(TaskContext& context)
{
  choose_template<VariantKind::GPU>(context);
}
}  // namespace cunumeric
