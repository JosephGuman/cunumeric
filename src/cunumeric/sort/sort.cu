#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "cunumeric/sort/sort.h"
#include "cunumeric/sort/sort_template.inl"

#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/execution_policy.h>

#include "cunumeric/cuda_help.h"

namespace cunumeric {

using namespace Legion;

template <LegateTypeCode CODE, int32_t DIM>
struct SortImplBody<VariantKind::GPU, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  void thrust_sort(const VAL* inptr, VAL* outptr, const size_t volume, const size_t sort_dim_size)
  {
    thrust::device_ptr<const VAL> dev_input_ptr(inptr);
    thrust::device_ptr<VAL> dev_output_ptr(outptr);
    thrust::copy(dev_input_ptr, dev_input_ptr + volume, dev_output_ptr);
    // same approach as cupy implemntation --> combine multiple individual sorts into single
    // kernel with data tuples - (id_sub-sort, actual_data)
    if (DIM == 1) {
      thrust::stable_sort(dev_output_ptr, dev_output_ptr + volume);
    } else {
      // in this case we know we are sorting for the *last* index
      const uint64_t max_elements_per_kernel =
        1 << 22;  // TODO check amount of available GPU memory from config
      const uint64_t number_sorts_per_kernel =
        std::max(1ul, std::min(volume, max_elements_per_kernel) / sort_dim_size);
      const uint64_t number_sorts = volume / sort_dim_size;

      // std::cout << "Number of sorts per kernel: " << number_sorts_per_kernel << std::endl;

      if (number_sorts_per_kernel >=
          32)  // key-tuple sort has quite some overhead -- only utilize if beneficial
      {
        // allocate memory for keys (iterating +=1 for each individual sort dimension)
        // ensure keys have minimal bit-length (needs values up to number_sorts_per_kernel-1)!
        // TODO!!!!
        auto keys_array = create_buffer<uint32_t>(number_sorts_per_kernel * sort_dim_size,
                                                  Legion::Memory::Kind::GPU_FB_MEM);
        thrust::device_ptr<uint32_t> dev_key_ptr(keys_array.ptr(0));

        for (uint64_t sort_part = 0; sort_part < number_sorts;
             sort_part += number_sorts_per_kernel) {
          // compute size of batch (might be smaller for the last call)
          const uint64_t num_elements =
            std::min(number_sorts - sort_part, max_elements_per_kernel) * sort_dim_size;
          const uint64_t offset = sort_part * sort_dim_size;

          // reinit keys
          thrust::transform(thrust::make_counting_iterator<uint64_t>(0),
                            thrust::make_counting_iterator<uint64_t>(num_elements),
                            thrust::make_constant_iterator<uint64_t>(sort_dim_size),
                            dev_key_ptr,
                            thrust::divides<uint64_t>());

          // sort
          auto combined =
            thrust::make_zip_iterator(thrust::make_tuple(dev_key_ptr, dev_output_ptr + offset));
          thrust::stable_sort(
            combined, combined + num_elements, thrust::less<thrust::tuple<size_t, VAL>>());
        }
      } else {
        // number_sorts_per_kernel too small ----> we sort one after another
        for (uint64_t sort_part = 0; sort_part < number_sorts; sort_part++) {
          const uint64_t offset = sort_part * sort_dim_size;
          thrust::stable_sort(dev_output_ptr + offset, dev_output_ptr + offset + sort_dim_size);
        }
      }
    }
  }

  void operator()(AccessorRO<VAL, DIM> input,
                  AccessorWO<VAL, DIM> output,
                  const Pitches<DIM - 1>& pitches,
                  const Rect<DIM>& rect,
                  const bool dense,
                  const size_t volume,
                  const Legion::DomainPoint global_shape,
                  const bool is_index_space,
                  const Legion::DomainPoint index_point,
                  const Legion::Domain domain)
  {
#ifdef DEBUG_CUNUMERIC
    std::cout << "GPU(" << index_point[0] << "): local size = " << volume
              << ", dist. = " << is_index_space << ", index_point = " << index_point
              << ", domain/volume = " << domain << "/" << domain.get_volume()
              << ", dense = " << dense << std::endl;
#endif
    const size_t sort_dim_size = global_shape[DIM - 1];
    assert(!is_index_space || DIM > 1);  // not implemented for now
    if (dense) {
      thrust_sort(input.ptr(rect), output.ptr(rect), volume, sort_dim_size);
    } else {
      // compute contiguous memory block
      int contiguous_elements = 1;
      for (int i = DIM - 1; i >= 0; i--) {
        auto diff = 1 + rect.hi[i] - rect.lo[i];
        contiguous_elements *= diff;
        if (diff < global_shape[i]) { break; }
      }

      uint64_t elements_processed = 0;
      while (elements_processed < volume) {
        Legion::Point<DIM> start_point = pitches.unflatten(elements_processed, rect.lo);
        thrust_sort(
          input.ptr(start_point), output.ptr(start_point), contiguous_elements, sort_dim_size);
        elements_processed += contiguous_elements;
      }
    }
  }
};

/*static*/ void SortTask::gpu_variant(TaskContext& context)
{
  sort_template<VariantKind::GPU>(context);
}

}  // namespace cunumeric
