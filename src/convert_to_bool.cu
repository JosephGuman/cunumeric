#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "convert.h"

using namespace Legion;

// instantiate Convert's tasks' gpu variants
// we omit the T1 == T2 case

// T == bool
namespace legate {
namespace numpy {
template void PointTask<ConvertTask<bool, __half>>::gpu_variant(const Task*,
                                                                const std::vector<PhysicalRegion>&,
                                                                Context,
                                                                Runtime*);
template void PointTask<ConvertTask<bool, float>>::gpu_variant(const Task*,
                                                               const std::vector<PhysicalRegion>&,
                                                               Context,
                                                               Runtime*);
template void PointTask<ConvertTask<bool, double>>::gpu_variant(const Task*,
                                                                const std::vector<PhysicalRegion>&,
                                                                Context,
                                                                Runtime*);
template void PointTask<ConvertTask<bool, int16_t>>::gpu_variant(const Task*,
                                                                 const std::vector<PhysicalRegion>&,
                                                                 Context,
                                                                 Runtime*);
template void PointTask<ConvertTask<bool, int32_t>>::gpu_variant(const Task*,
                                                                 const std::vector<PhysicalRegion>&,
                                                                 Context,
                                                                 Runtime*);
template void PointTask<ConvertTask<bool, int64_t>>::gpu_variant(const Task*,
                                                                 const std::vector<PhysicalRegion>&,
                                                                 Context,
                                                                 Runtime*);
template void PointTask<ConvertTask<bool, uint16_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<bool, uint32_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<bool, uint64_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
}  // namespace numpy
}  // namespace legate
