#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "unary/scalar_unary_red.h"
#include "unary/unary_red_util.h"
#include "core.h"
#include "cuda_help.h"
#include "deserializer.h"
#include "dispatch.h"
#include "point_task.h"

#include <thrust/reduce.h>

namespace legate {
namespace numpy {

using namespace Legion;

namespace gpu {

template <typename Op,
          typename Output,
          typename ReadAcc,
          typename Pitches,
          typename Point,
          typename VAL>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  reduction_kernel(size_t volume,
                   Op op,
                   Output result,
                   ReadAcc in,
                   Pitches pitches,
                   Point origin,
                   size_t iters,
                   VAL identity)
{
  auto value = identity;
  for (size_t idx = 0; idx < iters; idx++) {
    const size_t offset = (idx * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (offset < volume) {
      auto point = pitches.unflatten(offset, origin);
      Op::template fold<true>(value, in[point]);
    }
  }
  // Every thread in the thread block must participate in the exchange to get correct results
  reduce_output(result, value);
}

template <UnaryRedCode OP_CODE>
struct ScalarUnaryRedImpl {
  template <LegateTypeCode CODE,
            int DIM,
            std::enable_if_t<UnaryRedOp<OP_CODE, CODE>::valid> * = nullptr>
  UntypedScalar operator()(Shape &shape, RegionField &in_rf)
  {
    using OP  = UnaryRedOp<OP_CODE, CODE>;
    using VAL = legate_type_of<CODE>;

    auto rect = shape.to_rect<DIM>();

    Pitches<DIM - 1> pitches;
    size_t volume = pitches.flatten(rect);

    if (volume == 0) return UntypedScalar(OP::identity);

    hipStream_t stream;
    hipStreamCreate(&stream);

    auto in             = in_rf.read_accessor<VAL, DIM>();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    DeferredReduction<typename OP::OP> result;
    size_t shmem_size = THREADS_PER_BLOCK / 32 * sizeof(VAL);

    if (blocks >= MAX_REDUCTION_CTAS) {
      const size_t iters = (blocks + MAX_REDUCTION_CTAS - 1) / MAX_REDUCTION_CTAS;
      reduction_kernel<<<MAX_REDUCTION_CTAS, THREADS_PER_BLOCK, shmem_size, stream>>>(
        volume, typename OP::OP{}, result, in, pitches, rect.lo, iters, OP::identity);
    } else
      reduction_kernel<<<blocks, THREADS_PER_BLOCK, shmem_size, stream>>>(
        volume, typename OP::OP{}, result, in, pitches, rect.lo, 1, OP::identity);

    // TODO: We eventually want to unblock this step
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    return UntypedScalar(result.read());
  }

  template <LegateTypeCode CODE,
            int DIM,
            std::enable_if_t<!UnaryRedOp<OP_CODE, CODE>::valid> * = nullptr>
  UntypedScalar operator()(Shape &shape, RegionField &in_rf)
  {
    assert(false);
    return UntypedScalar();
  }
};

struct ScalarUnaryRedDispatch {
  template <UnaryRedCode OP_CODE>
  UntypedScalar operator()(Shape &shape, RegionField &in)
  {
    return double_dispatch(in.dim(), in.code(), ScalarUnaryRedImpl<OP_CODE>{}, shape, in);
  }
};

}  // namespace gpu

/*static*/ UntypedScalar ScalarUnaryRedTask::gpu_variant(const Task *task,
                                                         const std::vector<PhysicalRegion> &regions,
                                                         Context context,
                                                         Runtime *runtime)
{
  Deserializer ctx(task, regions);

  UnaryRedCode op_code;
  Shape shape;
  RegionField in;

  deserialize(ctx, op_code);
  deserialize(ctx, shape);
  deserialize(ctx, in);

  return op_dispatch(op_code, gpu::ScalarUnaryRedDispatch{}, shape, in);
}

}  // namespace numpy
}  // namespace legate
