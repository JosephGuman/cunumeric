#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "unary/scalar_unary_red.h"
#include "unary/scalar_unary_red_template.inl"

#include "cuda_help.h"

namespace legate {
namespace numpy {

using namespace Legion;

template <typename Op,
          typename Output,
          typename ReadAcc,
          typename Pitches,
          typename Point,
          typename VAL>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  reduction_kernel(size_t volume,
                   Op op,
                   Output out,
                   ReadAcc in,
                   Pitches pitches,
                   Point origin,
                   size_t iters,
                   VAL identity)
{
  auto value = identity;
  for (size_t idx = 0; idx < iters; idx++) {
    const size_t offset = (idx * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (offset < volume) {
      auto point = pitches.unflatten(offset, origin);
      Op::template fold<true>(value, in[point]);
    }
  }
  // Every thread in the thread block must participate in the exchange to get correct results
  reduce_output(out, value);
}

template <typename Output, typename ReadAcc, typename Pitches, typename Point, typename VAL>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM) contains_kernel(
  size_t volume, Output out, ReadAcc in, Pitches pitches, Point origin, size_t iters, VAL to_find)
{
  bool value = false;
  for (size_t idx = 0; idx < iters; idx++) {
    const size_t offset = (idx * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (offset < volume) {
      auto point = pitches.unflatten(offset, origin);
      SumReduction<bool>::fold<true>(value, in[point] == to_find);
    }
  }
  // Every thread in the thread block must participate in the exchange to get correct results
  reduce_output(out, value);
}

template <typename Output, typename Pitches, typename Point, typename VAL, int32_t DIM>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM) count_nonzero_kernel(
  size_t volume, Output out, AccessorRO<VAL, DIM> in, Pitches pitches, Point origin, size_t iters)
{
  uint64_t value = 0;
  for (size_t idx = 0; idx < iters; idx++) {
    const size_t offset = (idx * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (offset < volume) {
      auto point = pitches.unflatten(offset, origin);
      SumReduction<uint64_t>::fold<true>(value, in[point] != VAL(0));
    }
  }
  // Every thread in the thread block must participate in the exchange to get correct results
  reduce_output(out, value);
}

template <UnaryRedCode OP_CODE, LegateTypeCode CODE, int DIM>
struct ScalarUnaryRedImplBody<VariantKind::GPU, OP_CODE, CODE, DIM> {
  using OP    = UnaryRedOp<OP_CODE, CODE>;
  using LG_OP = typename OP::OP;
  using VAL   = legate_type_of<CODE>;

  void operator()(OP func,
                  VAL& result,
                  AccessorRO<VAL, DIM> in,
                  const Rect<DIM>& rect,
                  const Pitches<DIM - 1>& pitches,
                  bool dense) const
  {
    hipStream_t stream;
    hipStreamCreate(&stream);

    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    DeferredReduction<typename OP::OP> out;
    size_t shmem_size = THREADS_PER_BLOCK / 32 * sizeof(VAL);

    if (blocks >= MAX_REDUCTION_CTAS) {
      const size_t iters = (blocks + MAX_REDUCTION_CTAS - 1) / MAX_REDUCTION_CTAS;
      reduction_kernel<<<MAX_REDUCTION_CTAS, THREADS_PER_BLOCK, shmem_size, stream>>>(
        volume, typename OP::OP{}, out, in, pitches, rect.lo, iters, LG_OP::identity);
    } else
      reduction_kernel<<<blocks, THREADS_PER_BLOCK, shmem_size, stream>>>(
        volume, typename OP::OP{}, out, in, pitches, rect.lo, 1, LG_OP::identity);

    // TODO: We eventually want to unblock this step
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    result = out.read();
  }
};

template <LegateTypeCode CODE, int DIM>
struct ScalarUnaryRedImplBody<VariantKind::GPU, UnaryRedCode::CONTAINS, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  void operator()(bool& result,
                  AccessorRO<VAL, DIM> in,
                  const UntypedScalar& to_find_scalar,
                  const Rect<DIM>& rect,
                  const Pitches<DIM - 1>& pitches,
                  bool dense) const
  {
    hipStream_t stream;
    hipStreamCreate(&stream);

    const auto to_find  = to_find_scalar.value<VAL>();
    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    DeferredReduction<SumReduction<bool>> out;
    size_t shmem_size = THREADS_PER_BLOCK / 32 * sizeof(bool);

    if (blocks >= MAX_REDUCTION_CTAS) {
      const size_t iters = (blocks + MAX_REDUCTION_CTAS - 1) / MAX_REDUCTION_CTAS;
      contains_kernel<<<MAX_REDUCTION_CTAS, THREADS_PER_BLOCK, shmem_size, stream>>>(
        volume, out, in, pitches, rect.lo, iters, to_find);
    } else
      contains_kernel<<<blocks, THREADS_PER_BLOCK, shmem_size, stream>>>(
        volume, out, in, pitches, rect.lo, 1, to_find);

    // TODO: We eventually want to unblock this step
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    result = out.read();
  }
};

template <LegateTypeCode CODE, int DIM>
struct ScalarUnaryRedImplBody<VariantKind::GPU, UnaryRedCode::COUNT_NONZERO, CODE, DIM> {
  using VAL = legate_type_of<CODE>;

  void operator()(uint64_t& result,
                  AccessorRO<VAL, DIM> in,
                  const Rect<DIM>& rect,
                  const Pitches<DIM - 1>& pitches,
                  bool dense) const
  {
    hipStream_t stream;
    hipStreamCreate(&stream);

    const size_t volume = rect.volume();
    const size_t blocks = (volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    DeferredReduction<SumReduction<uint64_t>> out;
    size_t shmem_size = THREADS_PER_BLOCK / 32 * sizeof(uint64_t);

    if (blocks >= MAX_REDUCTION_CTAS) {
      const size_t iters = (blocks + MAX_REDUCTION_CTAS - 1) / MAX_REDUCTION_CTAS;
      count_nonzero_kernel<<<MAX_REDUCTION_CTAS, THREADS_PER_BLOCK, shmem_size, stream>>>(
        volume, out, in, pitches, rect.lo, iters);
    } else
      count_nonzero_kernel<<<blocks, THREADS_PER_BLOCK, shmem_size, stream>>>(
        volume, out, in, pitches, rect.lo, 1);

    // TODO: We eventually want to unblock this step
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    result = out.read();
  }
};

/*static*/ UntypedScalar ScalarUnaryRedTask::gpu_variant(TaskContext& context)
{
  return scalar_unary_red_template<VariantKind::GPU>(context);
}

}  // namespace numpy
}  // namespace legate
