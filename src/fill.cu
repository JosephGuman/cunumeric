#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "fill.h"

using namespace Legion;

namespace legate {
namespace numpy {

template void PointTask<FillTask<__half>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<float>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<double>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<int16_t>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<int32_t>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<int64_t>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<uint16_t>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<uint32_t>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<uint64_t>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<bool>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<complex<float>>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<FillTask<complex<double>>>::gpu_variant(const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
}    // namespace numpy
}    // namespace legate
