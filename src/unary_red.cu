#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include <iostream>

#include "unary_red.h"
#include "unary_red_util.h"
#include "core.h"
#include "deserializer.h"
#include "dispatch.h"
#include "point_task.h"

namespace legate {
namespace numpy {

using namespace Legion;

namespace gpu {

template <typename T>
constexpr T div_and_ceil(T value, T divider)
{
  return std::max<T>((value + divider - 1) / divider, 1);
}

static constexpr coord_t WARP_SIZE = 32;

// This helper class is to compute the shape of thread blocks for reduction kernels.
// The strategy is to parallelize on dimensions, from the outermost one to the innermost,
// that are not being collapsed, thereby having threads work on independet lanes of
// reductions as much as possible. In case where the non-collapsing dimensions don't
// have enough elements to be assigned to the threads, we also parallelize on
// the collapsing domain. One exceptional case to this strategy is where the collapsing
// dimension is the innermost one, in which case we prefer that dimension to the others
// in order to enjoy wrap coalescing. The maximum degree of such parallelism woudl be 32,
// which is the size of a wrap.
template <int32_t DIM>
struct ThreadBlock {
  void initialize(const Rect<DIM> &domain, int32_t collapsed_dim)
  {
    auto remaining = static_cast<coord_t>(THREADS_PER_BLOCK);

    Point<DIM> domain_extents;
    for (int32_t idx = 0; idx < DIM; ++idx)
      domain_extents[idx] = domain.hi[idx] - domain.lo[idx] + 1;

    // If the innermost dimension is being collapsed, we assign at least one warp to it
    // for warp coalsecing.
    if (collapsed_dim == DIM - 1) {
      auto extent             = std::min<coord_t>(WARP_SIZE, domain_extents[collapsed_dim]);
      extents_[collapsed_dim] = extent;
      remaining               = std::max<coord_t>(remaining / extent, 1);
    }

    // Then, we compute how many threads there should be along aech dimension,
    // excluding the one being collapsed
    for (int32_t idx = DIM - 1; idx >= 0; --idx) {
      if (idx == collapsed_dim) continue;
      auto extent   = std::min(remaining, domain_extents[idx]);
      extents_[idx] = extent;
      remaining     = std::max<coord_t>(remaining / extent, 1);
    }

    // Finally, we determine degree of parallelism for the collapsed dimension if we didn't above
    if (collapsed_dim != DIM - 1)
      extents_[collapsed_dim] = std::min(remaining, domain_extents[collapsed_dim]);

    // Cache the aggregate number of threads per increment in each dimension,
    // which later will be used for de-linearization of a thread id
    num_threads_ = 1;
    for (int32_t idx = DIM - 1; idx >= 0; --idx) {
      pitches_[idx] = num_threads_;
      num_threads_ *= extents_[idx];
    }
  }

  // Compute a relative coordiate of a given thread
  __host__ __device__ Point<DIM> point(coord_t tid)
  {
    Point<DIM> p;
    for (int32_t dim = 0; dim < DIM; ++dim) {
      p[dim] = tid / pitches_[dim];
      tid    = tid % pitches_[dim];
    }
    return p;
  }

  // Total number of threads
  size_t num_threads_;
  // Number of threads along each dimension
  Point<DIM> extents_;
  // Aggregate number of threads per increment in each dimension
  Point<DIM> pitches_;
};

// This class represents a set of concurrent thread blocks. Concurrent thread blocks form
// hyperplanes in N-dimensional integer lattice such that the collapsed dimension is normal to them.
// The size of thread blocks is determined by the maximum number of CTAs for a given kernel;
// the number of concurrent thread blocks is the minimum number of hyperplanes whose aggregate
// volume exceeds the maximum number of CTAs.
template <int32_t DIM>
struct ThreadBlocks {
  void initialize(const Rect<DIM> &domain, int32_t collapsed_dim)
  {
    collapsed_dim_ = collapsed_dim;
    block_.initialize(domain, collapsed_dim);

    for (int32_t idx = 0; idx < DIM; ++idx) {
      auto domain_extent = domain.hi[idx] - domain.lo[idx] + 1;
      extents_[idx]      = div_and_ceil(domain_extent, block_.extents_[idx]);
    }

    // We want the collapsed dimension to be the outermost one when
    // de-linearizing the block id.
    dim_order_[0] = collapsed_dim_;
    for (int32_t dim = 0, idx = 1; dim < DIM; ++dim)
      if (dim != collapsed_dim_) dim_order_[idx++] = dim;

    // Compute the aggregate number of blocks per increment in each dimension
    coord_t num_blocks = 1;
    for (int32_t idx = DIM - 1; idx >= 0; --idx) {
      auto dim      = dim_order_[idx];
      pitches_[dim] = num_blocks;
      num_blocks *= extents_[dim];
    }
    // For now we say all blocks can run concurrent.
    num_blocks_ = num_blocks;
    // Also compute the stride on the collapsed dimension
    collapsed_dim_stride_ = extents_[collapsed_dim_] * block_.extents_[collapsed_dim_];
  }

  // De-linearized the linearized block id and thread it into an N-dimensional point
  __host__ __device__ Point<DIM> point(coord_t bid, coord_t tid, const Point<DIM> &origin)
  {
    Point<DIM> p = origin;
    for (int32_t dim : dim_order_) {
      p[dim] += (bid / pitches_[dim]) * block_.extents_[dim];
      bid = bid % pitches_[dim];
    }
    p += block_.point(tid);
    return p;
  }

  void compute_maximum_concurrency(const void *func)
  {
    int32_t num_ctas = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_ctas, func, num_threads(), 0);

    size_t plane_size = pitches_[collapsed_dim_];
    // Calculate the number of planes whose volume barely exceeds the maximum number of CTAs
    size_t max_num_concurrent_planes =
      std::max<size_t>(div_and_ceil<size_t>(num_ctas, plane_size), 1);
    // Then we update the number of concurrent thread blocks and the stride on the collapsed
    // dimension
    num_blocks_           = plane_size * max_num_concurrent_planes;
    collapsed_dim_stride_ = max_num_concurrent_planes * block_.extents_[collapsed_dim_];
  }

  __host__ __device__ inline void next_point(Point<DIM> &point) const
  {
    point[collapsed_dim_] += collapsed_dim_stride_;
  }

  constexpr size_t num_blocks() const { return num_blocks_; }
  constexpr size_t num_threads() const { return block_.num_threads_; }

  // List of dimensions, from the outermost one to the innermost
  int32_t dim_order_[DIM];
  int32_t collapsed_dim_;
  coord_t collapsed_dim_stride_;
  // Shape of each thread block
  ThreadBlock<DIM> block_;
  // Number of thread blocks along each dimension
  Point<DIM> extents_;
  // Aggregate number of thread blocks per increment in each dimension
  Point<DIM> pitches_;
  // Number of concurrent thread blocks
  size_t num_blocks_;
};

template <int32_t DIM>
std::ostream &operator<<(std::ostream &os, const ThreadBlock<DIM> &block)
{
  os << "ThreadBlock(extents: " << block.extents_ << ", pitches: " << block.pitches_ << ")";
  return os;
}

template <int32_t DIM>
std::ostream &operator<<(std::ostream &os, const ThreadBlocks<DIM> &blocks)
{
  os << "ThreadBlocks(" << blocks.block_ << ", extents: " << blocks.extents_
     << ", pitches: " << blocks.pitches_ << ", num concurrent blocks: " << blocks.num_blocks_
     << ", dim order: {";
  for (int32_t dim : blocks.dim_order_) os << dim << ", ";
  os << "})";

  return os;
}

template <typename T>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  dense_init_kernel(size_t volume, T *out, T init)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  out[idx] = init;
}

template <typename WriteAcc, typename T, typename Pitches, typename Point>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  generic_init_kernel(size_t volume, WriteAcc out, T init, Pitches pitches, Point lo)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, lo);
  out[point] = init;
}

template <typename Op, typename VAL, int32_t DIM>
static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  red_kernel(Op op,
             AccessorWO<VAL, DIM> out,
             AccessorRO<VAL, DIM> in,
             VAL identity,
             ThreadBlocks<DIM> blocks,
             Rect<DIM> domain,
             int32_t collapsed_dim)
{
  coord_t tid      = threadIdx.x;
  coord_t bid      = blockIdx.x;
  Point<DIM> point = blocks.point(bid, tid, domain.lo);
  if (!domain.contains(point)) return;

  auto result = identity;
  while (point[collapsed_dim] <= domain.hi[collapsed_dim]) {
    Op::template fold<true>(result, in[point]);
    blocks.next_point(point);
  }

  if (result != identity) Op::template fold<false>(out[point], result);
}

template <UnaryRedCode OP_CODE>
struct UnaryRedImpl {
  template <LegateTypeCode CODE,
            int32_t RHS_DIM,
            std::enable_if_t<(RHS_DIM > 1) && UnaryRedOp<OP_CODE, CODE>::valid> * = nullptr>
  void operator()(int32_t collapsed_dim,
                  Shape &lhs_shape,
                  Shape &rhs_shape,
                  RegionField &lhs_init_rf,
                  RegionField &lhs_red_rf,
                  RegionField &rhs_rf)
  {
    constexpr int32_t LHS_DIM = RHS_DIM - 1;
    using OP                  = UnaryRedOp<OP_CODE, CODE>;
    using VAL                 = legate_type_of<CODE>;

    Pitches<LHS_DIM - 1> lhs_pitches;
    auto lhs_rect     = lhs_shape.to_rect<LHS_DIM>();
    size_t lhs_volume = lhs_pitches.flatten(lhs_rect);

    if (lhs_volume == 0) return;

    auto lhs_init = lhs_init_rf.write_accessor<VAL, LHS_DIM>();
#ifndef LEGION_BOUNDS_CHECKS
    // Check to see if this is dense or not
    bool dense = lhs_init.accessor.is_dense_row_major(lhs_rect);
#else
    // No dense execution if we're doing bounds checks
    bool dense = false;
#endif

    const size_t lhs_blocks = (lhs_volume + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    if (dense) {
      auto lhs = lhs_init.ptr(lhs_rect);
      dense_init_kernel<<<lhs_blocks, THREADS_PER_BLOCK>>>(lhs_volume, lhs, OP::identity);
    } else {
      generic_init_kernel<<<lhs_blocks, THREADS_PER_BLOCK>>>(
        lhs_volume, lhs_init, OP::identity, lhs_pitches, lhs_rect.lo);
    }

    ThreadBlocks<RHS_DIM> blocks;
    auto rhs_rect = rhs_shape.to_rect<RHS_DIM>();
    blocks.initialize(rhs_rect, collapsed_dim);
    blocks.compute_maximum_concurrency(
      reinterpret_cast<const void *>(red_kernel<OP, VAL, RHS_DIM>));
    auto lhs_red = lhs_red_rf.write_accessor<VAL, RHS_DIM>();
    auto rhs     = rhs_rf.read_accessor<VAL, RHS_DIM>();

    red_kernel<<<blocks.num_blocks(), blocks.num_threads()>>>(
      OP{}, lhs_red, rhs, OP::identity, blocks, rhs_rect, collapsed_dim);
  }

  template <LegateTypeCode CODE,
            int32_t RHS_DIM,
            std::enable_if_t<RHS_DIM <= 1 || !UnaryRedOp<OP_CODE, CODE>::valid> * = nullptr>
  void operator()(int32_t collapsed_dim,
                  Shape &lhs_shape,
                  Shape &rhs_shape,
                  RegionField &lhs_init,
                  RegionField &lhs_red,
                  RegionField &rhs)
  {
    assert(false);
  }
};

struct UnaryRedDispatch {
  template <UnaryRedCode OP_CODE>
  void operator()(int32_t collapsed_dim,
                  Shape &lhs_shape,
                  Shape &rhs_shape,
                  RegionField &lhs_init,
                  RegionField &lhs_red,
                  RegionField &rhs)
  {
    return double_dispatch(rhs.dim(),
                           rhs.code(),
                           UnaryRedImpl<OP_CODE>{},
                           collapsed_dim,
                           lhs_shape,
                           rhs_shape,
                           lhs_init,
                           lhs_red,
                           rhs);
  }
};

}  // namespace gpu

/*static*/ void UnaryRedTask::gpu_variant(const Task *task,
                                          const std::vector<PhysicalRegion> &regions,
                                          Context context,
                                          Runtime *runtime)
{
  Deserializer ctx(task, regions);

  int32_t collapsed_dim;
  UnaryRedCode op_code;
  Shape lhs_shape;
  Shape rhs_shape;
  // out_init and out_red are aliases of the same region field but with different transformations
  RegionField lhs_init;
  RegionField lhs_red;
  RegionField rhs;

  deserialize(ctx, collapsed_dim);
  deserialize(ctx, op_code);
  deserialize(ctx, lhs_shape);
  deserialize(ctx, rhs_shape);
  deserialize(ctx, lhs_init);
  deserialize(ctx, lhs_red);
  deserialize(ctx, rhs);

  return op_dispatch(
    op_code, gpu::UnaryRedDispatch{}, collapsed_dim, lhs_shape, rhs_shape, lhs_init, lhs_red, rhs);
}

}  // namespace numpy
}  // namespace legate
