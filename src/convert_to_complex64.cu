#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "convert.h"

using namespace Legion;

// instantiate Convert's tasks' gpu variants
// we omit the T1 == T2 case

// To == complex<float>
namespace legate {
namespace numpy {
template void PointTask<ConvertTask<complex<float>, __half>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, float>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, double>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, int16_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, int32_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, int64_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, uint16_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, uint32_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, uint64_t>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, bool>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
template void PointTask<ConvertTask<complex<float>, complex<double>>>::gpu_variant(
  const Task*, const std::vector<PhysicalRegion>&, Context, Runtime*);
}  // namespace numpy
}  // namespace legate
